#include "hip/hip_runtime.h"
#include <gauge_field.h>
#include <gauge_field_order.h>
#include <color_spinor_field.h>
#include <color_spinor_field_order.h>
#include <index_helper.cuh>
#include <stencil.h>
#include <color_spinor.h>
#include <tune_quda.h>
#include <worker.h>

/**
   This is the covariant derivative based on the basic gauged Laplace operator
*/

namespace quda {

#ifdef GPU_CONTRACT

  /**
     @brief Parameter structure for driving the covariant derivative
   */
  template <typename Float, int nSpin, int nColor, QudaReconstructType reconstruct>
  struct CovDevArg {
    typedef typename colorspinor_mapper<Float,nSpin,nColor>::type F;
    typedef typename gauge_mapper<Float,reconstruct>::type G;

    F out;                // output vector field
    const F in;           // input vector field
    const G U;            // the gauge field
    const int parity;     // only use this for single parity fields
    const int nParity;    // number of parities we're working on
    const int nFace;      // hard code to 1 for now
    const int dim[5];     // full lattice dimensions
    const int commDim[4]; // whether a given dimension is partitioned or not
    const int volumeCB;   // checkerboarded volume
    const int mu;         // direction of the covariant derivative

    CovDevArg(ColorSpinorField &out, const ColorSpinorField &in, const GaugeField &U, const int parity, const int mu)
      : out(out), in(in), U(U), parity(parity), mu(mu), nParity(in.SiteSubset()), nFace(1),
	dim{ (3-nParity) * in.X(0), in.X(1), in.X(2), in.X(3), 1 },
      commDim{comm_dim_partitioned(0), comm_dim_partitioned(1), comm_dim_partitioned(2), comm_dim_partitioned(3)},
      volumeCB(in.VolumeCB())
    {
      if (!U.isNative())
      errorQuda("Unsupported field order colorspinor=%d gauge=%d combination\n", in.FieldOrder(), U.FieldOrder());
    }
  };

  /**
     Applies the off-diagonal part of the Laplace operator

     @param[out] out The out result field
     @param[in] U The gauge field
     @param[in] kappa Kappa value
     @param[in] in The input field
     @param[in] parity The site parity
     @param[in] x_cb The checkerboarded site index
   */
  template <typename Float, int nDim, int nColor, int mu, typename Vector, typename Arg>
  __device__ __host__ inline void applyCovDev(Vector &out, Arg &arg, int x_cb, int parity) {
    typedef Matrix<complex<Float>,nColor> Link;
    const int their_spinor_parity = (arg.nParity == 2) ? 1-parity : 0;

    int coord[5];
    getCoords(coord, x_cb, arg.dim, parity);
    coord[4] = 0;

    const int d = mu%4;

    if (mu < 4) {
      //Forward gather - compute fwd offset for vector fetch
      const int fwd_idx = linkIndexP1(coord, arg.dim, d);

      if ( arg.commDim[d] && (coord[d] + arg.nFace >= arg.dim[d]) ) {
	const int ghost_idx = ghostFaceIndex<1>(coord, arg.dim, d, arg.nFace);

	const Link U = arg.U(d, x_cb, parity);
	const Vector in = arg.in.Ghost(d, 1, ghost_idx, their_spinor_parity);

	out += U * in;
      } else {

	const Link U = arg.U(d, x_cb, parity);
	const Vector in = arg.in(fwd_idx, their_spinor_parity);

	out += U * in;
      }
    } else {
      //Backward gather - compute back offset for spinor and gauge fetch
      const int back_idx = linkIndexM1(coord, arg.dim, d);
      const int gauge_idx = back_idx;

      if ( arg.commDim[d] && (coord[d] - arg.nFace < 0) ) {
	const int ghost_idx = ghostFaceIndex<0>(coord, arg.dim, d, arg.nFace);

	const Link U = arg.U.Ghost(d, ghost_idx, 1-parity);
	const Vector in = arg.in.Ghost(d, 0, ghost_idx, their_spinor_parity);

	out += conj(U) * in;
      } else {
	
	const Link U = arg.U(d, gauge_idx, 1-parity);
	const Vector in = arg.in(back_idx, their_spinor_parity);

	out += conj(U) * in;
      }
    } // Forward/backward derivative

  }


  //out(x) = M*in
  template <typename Float, int nDim, int nSpin, int nColor, typename Arg>
  __device__ __host__ inline void covDev(Arg &arg, int x_cb, int parity)
  {
    typedef ColorSpinor<Float,nColor,nSpin> Vector;
    Vector out;

    switch (arg.mu) {
    case 0: applyCovDev<Float,nDim,nColor,0>(out, arg, x_cb, parity); break;
    case 1: applyCovDev<Float,nDim,nColor,1>(out, arg, x_cb, parity); break;
    case 2: applyCovDev<Float,nDim,nColor,2>(out, arg, x_cb, parity); break;
    case 3: applyCovDev<Float,nDim,nColor,3>(out, arg, x_cb, parity); break;
    case 4: applyCovDev<Float,nDim,nColor,4>(out, arg, x_cb, parity); break;
    case 5: applyCovDev<Float,nDim,nColor,5>(out, arg, x_cb, parity); break;
    case 6: applyCovDev<Float,nDim,nColor,6>(out, arg, x_cb, parity); break;
    case 7: applyCovDev<Float,nDim,nColor,7>(out, arg, x_cb, parity); break;
    }
    arg.out(x_cb, parity) = out;
  }

  // CPU kernel for applying the Laplace operator to a vector
  template <typename Float, int nDim, int nSpin, int nColor, typename Arg>
  void covDevCPU(Arg arg)
  {

    for (int parity= 0; parity < arg.nParity; parity++) {
      // for full fields then set parity from loop else use arg setting
      parity = (arg.nParity == 2) ? parity : arg.parity;

      for (int x_cb = 0; x_cb < arg.volumeCB; x_cb++) { // 4-d volume
	covDev<Float,nDim,nSpin,nColor>(arg, x_cb, parity);
      } // 4-d volumeCB
    } // parity

  }

  // GPU Kernel for applying the Laplace operator to a vector
  template <typename Float, int nDim, int nSpin, int nColor, typename Arg>
  __global__ void covDevGPU(Arg arg)
  {
    int x_cb = blockIdx.x*blockDim.x + threadIdx.x;

    // for full fields set parity from y thread index else use arg setting
    int parity = blockDim.y*blockIdx.y + threadIdx.y;

    if (x_cb >= arg.volumeCB) return;
    if (parity >= arg.nParity) return;

    covDev<Float,nDim,nSpin,nColor>(arg, x_cb, parity);
  }

  template <typename Float, int nDim, int nSpin, int nColor, typename Arg>
  class CovDev : public TunableVectorY {

  protected:
    Arg &arg;
    const ColorSpinorField &meta;

    long long flops() const
    {
      return 8*nColor*nColor*arg.nParity*(long long)meta.VolumeCB();
    }
    long long bytes() const
    {
      return arg.out.Bytes() + arg.in.Bytes() + arg.nParity*arg.U.Bytes()*meta.VolumeCB();
    }
    bool tuneGridDim() const { return false; }
    unsigned int minThreads() const { return arg.volumeCB; }

  public:
    CovDev(Arg &arg, const ColorSpinorField &meta) : TunableVectorY(arg.nParity), arg(arg), meta(meta)
    {
      strcpy(aux, meta.AuxString());
#ifdef MULTI_GPU
      char comm[5];
      comm[0] = (arg.commDim[0] ? '1' : '0');
      comm[1] = (arg.commDim[1] ? '1' : '0');
      comm[2] = (arg.commDim[2] ? '1' : '0');
      comm[3] = (arg.commDim[3] ? '1' : '0');
      comm[4] = '\0';
      strcat(aux,",comm=");
      strcat(aux,comm);
#endif
    }
    virtual ~CovDev() { }

    void apply(const hipStream_t &stream) {
      if (meta.Location() == QUDA_CPU_FIELD_LOCATION) {
	covDevCPU<Float,nDim,nSpin,nColor>(arg);
      } else {
        TuneParam tp = tuneLaunch(*this, getTuning(), getVerbosity());
	covDevGPU<Float,nDim,nSpin,nColor> <<<tp.grid,tp.block,tp.shared_bytes,stream>>>(arg);
      }
    }

    TuneKey tuneKey() const { return TuneKey(meta.VolString(), typeid(*this).name(), aux); }
  };


  template <typename Float, int nColor, QudaReconstructType recon>
    void ApplyCovDev(ColorSpinorField &out, const ColorSpinorField &in, const GaugeField &U, int parity, int mu)
  {
    constexpr int nDim = 4;
    if (in.Nspin() == 1) {
      constexpr int nSpin = 1;
      CovDevArg<Float,nSpin,nColor,recon> arg(out, in, U, parity, mu);
      CovDev<Float,nDim,nSpin,nColor,CovDevArg<Float,nSpin,nColor,recon> > myCovDev(arg, in);
      myCovDev.apply(0);
    } else if (in.Nspin() == 4) {
      constexpr int nSpin = 4;
      CovDevArg<Float,nSpin,nColor,recon> arg(out, in, U, parity, mu);
      CovDev<Float,nDim,nSpin,nColor,CovDevArg<Float,nSpin,nColor,recon> > myCovDev(arg, in);
      myCovDev.apply(0);
    } else {
      errorQuda("Unsupported nSpin=%d", in.Nspin());
    }
  }

  // template on the gauge reconstruction
  template <typename Float, int nColor>
    void ApplyCovDev(ColorSpinorField &out, const ColorSpinorField &in, const GaugeField &U, int parity, int mu)
  {
    if (U.Reconstruct()== QUDA_RECONSTRUCT_NO) {
      ApplyCovDev<Float,nColor,QUDA_RECONSTRUCT_NO>(out, in, U, parity, mu);
    } else if (U.Reconstruct()== QUDA_RECONSTRUCT_12) {
      ApplyCovDev<Float,nColor,QUDA_RECONSTRUCT_12>(out, in, U, parity, mu);
    } else if (U.Reconstruct()== QUDA_RECONSTRUCT_8) {
      ApplyCovDev<Float,nColor,QUDA_RECONSTRUCT_8> (out, in, U, parity, mu);
    } else {
      errorQuda("Unsupported reconstruct type %d\n", U.Reconstruct());
    }
  }

  // template on the number of colors
  template <typename Float>
    void ApplyCovDev(ColorSpinorField &out, const ColorSpinorField &in, const GaugeField &U, int parity, int mu)
  {
    if (in.Ncolor() == 3) {
      ApplyCovDev<Float,3>(out, in, U, parity, mu);
    } else {
      errorQuda("Unsupported number of colors %d\n", U.Ncolor());
    }
  }

  // this is the Worker pointer that may have issue additional work
  // while we're waiting on communication to finish
  namespace dslash {
    extern Worker* aux_worker;
  }

#endif // GPU_CONTRACT

  //Apply the covariant derivative operator
  //out(x) = U_{\mu}(x)in(x+mu) for mu = 0...3
  //out(x) = U^\dagger_mu'(x-mu')in(x-mu') for mu = 4...7 and we set mu' = mu-4
  void ApplyCovDev(ColorSpinorField &out, const ColorSpinorField &in, const GaugeField &U, int parity, int mu)		    
  {
#ifdef GPU_CONTRACT
    if (in.V() == out.V()) errorQuda("Aliasing pointers");
    if (in.FieldOrder() != out.FieldOrder())
      errorQuda("Field order mismatch in = %d, out = %d", in.FieldOrder(), out.FieldOrder());
    
    // check all precision match
    checkPrecision(out, in, U);

    // check all locations match
    checkLocation(out, in, U);

    const int nFace = 1;
    in.exchangeGhost((QudaParity)(1-parity), nFace, 0); // last parameter is dummy

    if (dslash::aux_worker) dslash::aux_worker->apply(0);

    if (U.Precision() == QUDA_DOUBLE_PRECISION) {
      ApplyCovDev<double>(out, in, U, parity, mu);
    } else if (U.Precision() == QUDA_SINGLE_PRECISION) {
      ApplyCovDev<float>(out, in, U, parity, mu);
    } else {
      errorQuda("Unsupported precision %d\n", U.Precision());
    }

    in.bufferIndex = (1 - in.bufferIndex);
#else
    errorQuda("Contraction kernels have not been built");
#endif
  }

} // namespace quda
