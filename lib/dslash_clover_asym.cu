
#include <hip/hip_runtime.h>
#ifdef USE_LEGACY_DSLASH

#include <cstdlib>
#include <cstdio>
#include <string>
#include <iostream>
#include <typeinfo>

#include <color_spinor_field.h>
#include <clover_field.h>

// these control the Wilson-type actions
#ifdef GPU_WILSON_DIRAC
//#define DIRECT_ACCESS_LINK
//#define DIRECT_ACCESS_WILSON_SPINOR
//#define DIRECT_ACCESS_WILSON_ACCUM
//#define DIRECT_ACCESS_WILSON_INTER
//#define DIRECT_ACCESS_WILSON_PACK_SPINOR
//#define DIRECT_ACCESS_CLOVER
#endif // GPU_WILSON_DIRAC

#include <quda_internal.h>
#include <dslash_quda.h>
#include <dslash.h>
#include <sys/time.h>
#include <blas_quda.h>

#include <inline_ptx.h>
#include <dslash_policy.cuh>

namespace quda {

  namespace asym_clover {

#undef GPU_STAGGERED_DIRAC
#include <dslash_constants.h>
#include <dslash_textures.h>
#include <dslash_index.cuh>

    // Enable shared memory dslash for Fermi architecture
    //#define SHARED_WILSON_DSLASH
    //#define SHARED_8_BYTE_WORD_SIZE // 8-byte shared memory access

#ifdef GPU_CLOVER_DIRAC
#define DD_CLOVER 2
#include <wilson_dslash_def.h>    // Wilson Dslash kernels (including clover)
#undef DD_CLOVER
#endif

#ifndef DSLASH_SHARED_FLOATS_PER_THREAD
#define DSLASH_SHARED_FLOATS_PER_THREAD 0
#endif

#include <dslash_quda.cuh>

  } // end namespace asym_clover

  using namespace asym_clover;

#ifdef GPU_CLOVER_DIRAC
  template <typename sFloat, typename gFloat, typename cFloat>
  class AsymCloverDslashCuda : public SharedDslashCuda {

  protected:
    const FullClover &clover;

    unsigned int sharedBytesPerThread() const
    {
      if (dslashParam.kernel_type == INTERIOR_KERNEL) {
	int reg_size = (typeid(sFloat)==typeid(double2) ? sizeof(double) : sizeof(float));
	return DSLASH_SHARED_FLOATS_PER_THREAD * reg_size;
      } else {
	return 0;
      }
    }

  public:
    AsymCloverDslashCuda(cudaColorSpinorField *out, const GaugeField &gauge, const FullClover &clover,
			 const cudaColorSpinorField *in, const cudaColorSpinorField *x, const double a,
                         const int parity, const int dagger, const int *commOverride)
      : SharedDslashCuda(out, in, x, gauge, parity, dagger, commOverride), clover(clover)
    { 
      QudaPrecision clover_prec = bindCloverTex(clover, parity, dslashParam);
      if (in->Precision() != clover_prec) errorQuda("Mixing clover and spinor precision not supported");
      dslashParam.a = a;
      dslashParam.a_f = a;
      dslashParam.cl_stride = clover.stride;
      dslashParam.rho = clover.rho;
      dslashParam.rho_f = clover.rho;

      if (!x) errorQuda("Asymmetric clover dslash only defined for Xpay");
    }

    virtual ~AsymCloverDslashCuda() {
      unbindSpinorTex<sFloat>(in, out, x);
      unbindCloverTex(clover);
    }

    void apply(const cudaStream_t &stream)
    {
#ifdef SHARED_WILSON_DSLASH
      if (dslashParam.kernel_type == EXTERIOR_KERNEL_X) errorQuda("Shared dslash does not yet support X-dimension partitioning");
#endif
#ifndef USE_TEXTURE_OBJECTS
      if (dslashParam.kernel_type == INTERIOR_KERNEL) bindSpinorTex<sFloat>(in, out, x);
#endif // USE_TEXTURE_OBJECTS
      TuneParam tp = tuneLaunch(*this, getTuning(), getVerbosity());
      setParam();
      dslashParam.block[0] = tp.aux.x; dslashParam.block[1] = tp.aux.y; dslashParam.block[2] = tp.aux.z; dslashParam.block[3] = tp.aux.w;
      for (int i=0; i<4; i++) dslashParam.grid[i] = ( (i==0 ? 2 : 1) * in->X(i)) / dslashParam.block[i];
      ASYM_DSLASH(asymCloverDslash, tp.grid, tp.block, tp.shared_bytes, stream, dslashParam);
    }

    long long flops() const {
      int clover_flops = 504;
      long long flops = DslashCuda::flops();
      switch(dslashParam.kernel_type) {
      case EXTERIOR_KERNEL_X:
      case EXTERIOR_KERNEL_Y:
      case EXTERIOR_KERNEL_Z:
      case EXTERIOR_KERNEL_T:
      case EXTERIOR_KERNEL_ALL:
	break;
      case INTERIOR_KERNEL:
      case KERNEL_POLICY:
	// clover flops are done in the interior kernel
	flops += clover_flops * in->VolumeCB();
	break;
      }
      return flops;
    }

    long long bytes() const {
      bool isHalf = in->Precision() == sizeof(short) ? true : false;
      int clover_bytes = 72 * in->Precision() + (isHalf ? 2*sizeof(float) : 0);
      long long bytes = DslashCuda::bytes();
      switch(dslashParam.kernel_type) {
      case EXTERIOR_KERNEL_X:
      case EXTERIOR_KERNEL_Y:
      case EXTERIOR_KERNEL_Z:
      case EXTERIOR_KERNEL_T:
      case EXTERIOR_KERNEL_ALL:
	break;
      case INTERIOR_KERNEL:
      case KERNEL_POLICY:
	bytes += clover_bytes*in->VolumeCB();
	break;
      }

      return bytes;
    }

  };
#endif // GPU_CLOVER_DIRAC

  void asymCloverDslashCuda(cudaColorSpinorField *out, const cudaGaugeField &gauge, const FullClover &clover,
			    const cudaColorSpinorField *in, const int parity, const int dagger, 
			    const cudaColorSpinorField *x, const double &a, const int *commOverride,
			    TimeProfile &profile)
  {
#ifdef GPU_CLOVER_DIRAC
    const_cast<cudaColorSpinorField*>(in)->createComms(1);

    DslashCuda *dslash = nullptr;
    if (in->Precision() == QUDA_DOUBLE_PRECISION) {
      dslash = new AsymCloverDslashCuda<double2, double2, double2>(out, gauge, clover, in, x, a, parity, dagger, commOverride);
    } else if (in->Precision() == QUDA_SINGLE_PRECISION) {
      dslash = new AsymCloverDslashCuda<float4, float4, float4>(out, gauge, clover, in, x, a, parity, dagger, commOverride);
    } else if (in->Precision() == QUDA_HALF_PRECISION) {
      dslash = new AsymCloverDslashCuda<short4, short4, short4>(out, gauge, clover, in, x, a, parity, dagger, commOverride);
    }

    dslash::DslashPolicyTune<DslashCuda> dslash_policy(
        *dslash, const_cast<cudaColorSpinorField *>(in), in->Volume(), in->GhostFace(), profile);
    dslash_policy.apply(0);

    delete dslash;
#else
    errorQuda("Clover dslash has not been built");
#endif

  }

}

#endif
