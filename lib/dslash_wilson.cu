
#include <hip/hip_runtime.h>
#ifdef USE_LEGACY_DSLASH

#include <cstdlib>
#include <cstdio>
#include <string>
#include <iostream>
#include <typeinfo>

#include <color_spinor_field.h>
#include <clover_field.h>

// these control the Wilson-type actions
#ifdef GPU_WILSON_DIRAC
//#define DIRECT_ACCESS_LINK
//#define DIRECT_ACCESS_WILSON_SPINOR
//#define DIRECT_ACCESS_WILSON_ACCUM
//#define DIRECT_ACCESS_WILSON_INTER
//#define DIRECT_ACCESS_WILSON_PACK_SPINOR
//#define DIRECT_ACCESS_CLOVER
#endif // GPU_WILSON_DIRAC


#include <quda_internal.h>
#include <dslash_quda.h>
#include <dslash.h>
#include <sys/time.h>
#include <blas_quda.h>

#include <inline_ptx.h>
#include <dslash_policy.cuh>

namespace quda {

  namespace wilson {

#include <dslash_constants.h>
#include <dslash_textures.h>
#include <dslash_index.cuh>

  // Enable shared memory dslash for Fermi architecture
  //#define SHARED_WILSON_DSLASH
  //#define SHARED_8_BYTE_WORD_SIZE // 8-byte shared memory access

#ifdef GPU_WILSON_DIRAC
#define DD_CLOVER 0
#include <wilson_dslash_def.h>    // Wilson Dslash kernels (including clover)
#undef DD_CLOVER
#endif

#ifndef DSLASH_SHARED_FLOATS_PER_THREAD
#define DSLASH_SHARED_FLOATS_PER_THREAD 0
#endif

#include <dslash_quda.cuh>

  } // end namespace wilson


  using namespace wilson;

#ifdef GPU_WILSON_DIRAC
  template <typename sFloat, typename gFloat>
  class WilsonDslashCuda : public SharedDslashCuda {

  protected:
    unsigned int sharedBytesPerThread() const
    {
      if (dslashParam.kernel_type == INTERIOR_KERNEL) { // Interior kernels use shared memory for common iunput
	int reg_size = (typeid(sFloat)==typeid(double2) ? sizeof(double) : sizeof(float));
	return DSLASH_SHARED_FLOATS_PER_THREAD * reg_size;
      } else { // Exterior kernels use no shared memory
	return 0;
      }
    }

  public:
    WilsonDslashCuda(cudaColorSpinorField *out, const GaugeField &gauge, const cudaColorSpinorField *in,
		     const cudaColorSpinorField *x, const double a, const int parity, const int dagger,
                     const int *commOverride)
      : SharedDslashCuda(out, in, x, gauge, parity, dagger, commOverride)
    { 
      dslashParam.a = a;
      dslashParam.a_f = a;
    }

    virtual ~WilsonDslashCuda() {
      unbindSpinorTex<sFloat>(in, out, x);
    }

    void apply(const cudaStream_t &stream)
    {
#ifdef SHARED_WILSON_DSLASH
      if (dslashParam.kernel_type == EXTERIOR_KERNEL_X) errorQuda("Shared dslash does not yet support X-dimension partitioning");
#endif
#ifndef USE_TEXTURE_OBJECTS
      if (dslashParam.kernel_type == INTERIOR_KERNEL) bindSpinorTex<sFloat>(in, out, x);
#endif // USE_TEXTURE_OBJECTS
      TuneParam tp = tuneLaunch(*this, getTuning(), getVerbosity());
      setParam();
      dslashParam.block[0] = tp.aux.x; dslashParam.block[1] = tp.aux.y; dslashParam.block[2] = tp.aux.z; dslashParam.block[3] = tp.aux.w;
      for (int i=0; i<4; i++) dslashParam.grid[i] = ( (i==0 ? 2 : 1) * in->X(i)) / dslashParam.block[i];
      DSLASH(dslash, tp.grid, tp.block, tp.shared_bytes, stream, dslashParam);
    }

  };
#endif // GPU_WILSON_DIRAC

  // Wilson wrappers
  void wilsonDslashCuda(cudaColorSpinorField *out, const cudaGaugeField &gauge, const cudaColorSpinorField *in,
			const int parity, const int dagger, const cudaColorSpinorField *x, const double &k,
			const int *commOverride, TimeProfile &profile)
  {
#ifdef GPU_WILSON_DIRAC
    const_cast<cudaColorSpinorField*>(in)->createComms(1);

    DslashCuda *dslash = nullptr;
    if (in->Precision() == QUDA_DOUBLE_PRECISION) {
      dslash = new WilsonDslashCuda<double2, double2>(out, gauge, in, x, k, parity, dagger, commOverride);
    } else if (in->Precision() == QUDA_SINGLE_PRECISION) {
      dslash = new WilsonDslashCuda<float4, float4>(out, gauge, in, x, k, parity, dagger, commOverride);
    } else if (in->Precision() == QUDA_HALF_PRECISION) {
      dslash = new WilsonDslashCuda<short4, short4>(out, gauge, in, x, k, parity, dagger, commOverride);
    }

    dslash::DslashPolicyTune<DslashCuda> dslash_policy(
        *dslash, const_cast<cudaColorSpinorField *>(in), in->Volume(), in->GhostFace(), profile);
    dslash_policy.apply(0);

    delete dslash;
#else
    errorQuda("Wilson dslash has not been built");
#endif // GPU_WILSON_DIRAC

  }

}

#endif
