#include "hip/hip_runtime.h"
/* C. Kallidonis: GPU kernel to perform di-quark contractions.
 * This function takes as input two forward propagators, performs
 * the relevant contractions the stores the result in another 
 * propagator object.
 * November 2017
 */

#include <qlua_contract.h>
#include <tune_quda.h>

namespace quda {
  
  //-- Main calculation kernel
#define ContractQQ_macro(cntrIdx, A, B, C, D)				\
  __device__ __host__ inline void performContractQQ ## cntrIdx(ColorSpinor<QUDA_REAL,QUDA_Nc,QUDA_Ns> *out, \
							       ContractQQArg *arg, \
							       int x_cb, \
							       int pty) \
  {									\
									\
    int eps[3][3] = { { 0, 1, 2},					\
		      { 1, 2, 0},					\
		      { 2, 0, 1} };					\
    									\
    const int Ns = QUDA_Ns;						\
    const int Nc = QUDA_Nc;						\
    									\
    Vector In1[QUDA_NVEC_PROP];						\
    Vector In2[QUDA_NVEC_PROP];						\
    									\
    for(int i=0;i<arg->nVec;i++){					\
      In1[i] = arg->pIn1[i](x_cb, pty);					\
      In2[i] = arg->pIn2[i](x_cb, pty);					\
    }									\
    rotateVectorBasis(In1,QLUA_quda2qdp);				\
    rotateVectorBasis(In2,QLUA_quda2qdp);				\
    									\
    									\
    for(int p_a = 0; p_a < Nc; p_a++){					\
      int i1 = eps[p_a][0];						\
      int j1 = eps[p_a][1];						\
      int k1 = eps[p_a][2];						\
      for (int p_b = 0; p_b < Nc; p_b++){				\
	int i2 = eps[p_b][0];						\
    	int j2 = eps[p_b][1];						\
	int k2 = eps[p_b][2];						\
    	for (int a = 0; a < Ns; a++){					\
    	  for (int b = 0; b < Ns; b++){					\
	    complex<QUDA_REAL> accum = 0.0;				\
	    for (int c = 0; c < Ns; c++){				\
	      int idx11 = (B)+Ns*i2;					\
	      int idx12 = (B)+Ns*j2;					\
	      int idx21 = (D)+Ns*i2;					\
	      int idx22 = (D)+Ns*j2;					\
	      								\
   	      accum += In1[idx11]((A),i1) * In2[idx22]((C),j1);		\
	      accum -= In1[idx12]((A),i1) * In2[idx21]((C),j1);		\
	      accum -= In1[idx11]((A),j1) * In2[idx22]((C),i1);		\
	      accum += In1[idx12]((A),j1) * In2[idx21]((C),i1);		\
	    }								\
	    out[b+Ns*k1](a,k2) = accum;					\
	  }								\
	}								\
      }									\
    }									\
  } //-- function closes
  
  ContractQQ_macro(12, c,c,a,b);
  ContractQQ_macro(13, c,a,c,b);
  ContractQQ_macro(14, c,a,b,c);
  ContractQQ_macro(23, a,c,c,b);
  ContractQQ_macro(24, a,c,b,c);
  ContractQQ_macro(34, a,b,c,c);
#undef ContractQQ_macro

  
  __device__ __host__ inline void computeContractQQ(ContractQQArg *arg, int x_cb, int pty){
    
    Vector out[QUDA_NVEC_PROP];
    
    switch(arg->cntrID){
    case cntr12:
      performContractQQ12(out, arg, x_cb, pty);
      break;
    case cntr13:
      performContractQQ13(out, arg, x_cb, pty);
      break;
    case cntr14:
      performContractQQ14(out, arg, x_cb, pty);
      break;
    case cntr23:
      performContractQQ23(out, arg, x_cb, pty);
      break;
    case cntr24:
      performContractQQ24(out, arg, x_cb, pty);
      break;
    case cntr34:
      performContractQQ34(out, arg, x_cb, pty);
      break;
    case cntr_INVALID: // Added it just to avoid the compilation warning, check has already been made
      break;
    }    

    rotateVectorBasis(out, QLUA_qdp2quda);
        
    for(int ivec=0;ivec<arg->nVec;ivec++)
      arg->pOut[ivec](x_cb, pty) = out[ivec];
    
  } //-- function closes
  
  
  //-- CPU kernel for performing the contractions
  void ContractQQ_CPU(ContractQQArg arg){    
    for (int parity= 0; parity < arg.nParity; parity++){
      parity = (arg.nParity == 2) ? parity : arg.parity;
      
      for (int x_cb = 0; x_cb < arg.volumeCB; x_cb++){
	computeContractQQ(&arg,x_cb,parity);
      }
    }
  }

  //-- GPU kernel for performing the contractions
  __global__ void ContractQQ_GPU(ContractQQArg *arg_dev){
    
    int x_cb = blockIdx.x*blockDim.x + threadIdx.x;
    
    int parity = blockDim.y*blockIdx.y + threadIdx.y;
    
    if (x_cb >= arg_dev->volumeCB) return;
    if (parity >= arg_dev->nParity) return;
    parity = (arg_dev->nParity == 2) ? parity : arg_dev->parity;
    
    computeContractQQ(arg_dev, x_cb, parity);
  }    
  

  //-- Class definition
  class ContractQQ : public TunableVectorY {

  protected:
    ContractQQArg &arg;
    ContractQQArg *arg_dev;
    const ColorSpinorField &meta;
    
    long long flops() const{
      return QUDA_Nc*QUDA_Nc*QUDA_Ns*QUDA_Ns*QUDA_Ns*4*(3+2)*arg.nParity*(long long)meta.VolumeCB();
    }
    long long bytes() const{
      return meta.Bytes() + QUDA_Nc*QUDA_Ns*2*meta.Bytes();
    }
    
    bool tuneGridDim() const { return false; }
    unsigned int minThreads() const { return arg.volumeCB; }
    
  public:
  ContractQQ(ContractQQArg &arg, ContractQQArg *arg_dev, const ColorSpinorField &meta) : TunableVectorY(arg.nParity), arg(arg), arg_dev(arg_dev), meta(meta)
    {
      strcpy(aux, meta.AuxString());
      strcat(aux, comm_dim_partitioned_string());
    }
    virtual ~ContractQQ() { }

    void apply(const hipStream_t &stream) {
      if (meta.Location() == QUDA_CPU_FIELD_LOCATION) {
	ContractQQ_CPU(arg);
      } else {
        TuneParam tp = tuneLaunch(*this, getTuning(), getVerbosity());

	ContractQQ_GPU <<<tp.grid,tp.block,tp.shared_bytes,stream>>> (arg_dev);
      }
    }

    TuneKey tuneKey() const { return TuneKey(meta.VolString(), typeid(*this).name(), aux); }
  }; //-- Class definition


  
  //-- Top level function, called within interface-qlua
  void cudaContractQQ(ColorSpinorField **propOut, ColorSpinorField **propIn1, ColorSpinorField **propIn2, int parity, int Nc, int Ns, QQParam cQQParam){
    
    if(Nc != QUDA_Nc) errorQuda("cudaContractQQ: Supports only Ncolor = %d. Got Nc = %d\n", QUDA_Nc, Nc);
    if(Ns != QUDA_Ns) errorQuda("cudaContractQQ: Supports only Nspin = %d.  Got Ns = %d\n", QUDA_Ns, Ns);
    
    ContractQQArg arg(propOut, propIn1, propIn2, parity, cQQParam);    
    ContractQQArg *arg_dev;
    hipMalloc((void**)&(arg_dev), sizeof(ContractQQArg) );
    hipMemcpy(arg_dev, &arg, sizeof(ContractQQArg), hipMemcpyHostToDevice);
    
    ContractQQ contract(arg, arg_dev, *propIn1[0]);
    contract.apply(0);

    hipDeviceSynchronize();
    checkCudaError();
  }

  
} //-- namespace quda
