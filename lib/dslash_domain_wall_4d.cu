
#include <hip/hip_runtime.h>
#ifdef USE_LEGACY_DSLASH

#include <cstdlib>
#include <cstdio>
#include <string>
#include <iostream>
#include <typeinfo>

#include <color_spinor_field.h>
#include <clover_field.h>

// these control the Wilson-type actions
#ifdef GPU_WILSON_DIRAC
//#define DIRECT_ACCESS_LINK
//#define DIRECT_ACCESS_WILSON_SPINOR
//#define DIRECT_ACCESS_WILSON_ACCUM
//#define DIRECT_ACCESS_WILSON_INTER
//#define DIRECT_ACCESS_WILSON_PACK_SPINOR
//#define DIRECT_ACCESS_CLOVER
#endif // GPU_WILSON_DIRAC

#include <quda_internal.h>
#include <dslash_quda.h>
#include <dslash.h>
#include <sys/time.h>
#include <blas_quda.h>

#include <inline_ptx.h>
#include <dslash_policy.cuh>

namespace quda {

  namespace domainwall4d {

#undef GPU_STAGGERED_DIRAC
#include <dslash_constants.h>
#include <dslash_textures.h>
#include <dslash_index.cuh>

    // Enable shared memory dslash for Fermi architecture
    //#define SHARED_WILSON_DSLASH
    //#define SHARED_8_BYTE_WORD_SIZE // 8-byte shared memory access

#ifdef GPU_DOMAIN_WALL_DIRAC
#include <dw_dslash4_def.h>       // Dslash4 Domain Wall kernels
#include <dw_dslash5_def.h>       // Dslash5 Domain Wall kernels
#include <dw_dslash5inv_def.h>    // Dslash5inv Domain Wall kernels
#endif

#ifndef DSLASH_SHARED_FLOATS_PER_THREAD
#define DSLASH_SHARED_FLOATS_PER_THREAD 0
#endif

#include <dslash_quda.cuh>
  }

  using namespace domainwall4d;

#ifdef GPU_DOMAIN_WALL_DIRAC
  template <typename sFloat, typename gFloat>
  class DomainWallDslash4DPCCuda : public DslashCuda {

  private:
    const int DS_type;

    bool checkGrid(TuneParam &param) const {
      if (param.grid.x > (unsigned int)deviceProp.maxGridSize[0] || param.grid.y > (unsigned int)deviceProp.maxGridSize[1]) {
        warningQuda("Autotuner is skipping blockDim=(%u,%u,%u), gridDim=(%u,%u,%u) because lattice volume is too large",
                    param.block.x, param.block.y, param.block.z, param.grid.x, param.grid.y, param.grid.z);
        return false;
      } else {
        return true;
      }
    }

  protected:
    bool advanceBlockDim(TuneParam &param) const
    {
      const unsigned int max_shared = 16384; // FIXME: use deviceProp.sharedMemPerBlock;
      const int step[2] = { deviceProp.warpSize, 1 };
      bool advance[2] = { false, false };

      // first try to advance block.x
      param.block.x += step[0];
      //memory constraint
      if (param.block.x > (unsigned int)deviceProp.maxThreadsDim[0] ||
          sharedBytesPerThread()*param.block.x*param.block.y > max_shared) {
        advance[0] = false;
        param.block.x = step[0]; // reset block.x
      } else {
        advance[0] = true; // successfully advanced block.x
      }

      if (!advance[0]) {  // if failed to advance block.x, now try block.y
        param.block.y += step[1];

	//memory constraint
        if (param.block.y > (unsigned)in->X(4) ||
            sharedBytesPerThread()*param.block.x*param.block.y > max_shared) {
          advance[1] = false;
          param.block.y = step[1]; // reset block.y
        } else {
          advance[1] = true; // successfully advanced block.y
        }
      }

      //thread constraint
      if ( (advance[0] || advance[1]) && param.block.x*param.block.y*param.block.z <= (unsigned)deviceProp.maxThreadsPerBlock) {
        param.grid = dim3( (dslashParam.threads+param.block.x-1) / param.block.x, 
            (in->X(4)+param.block.y-1) / param.block.y, 1);

        bool advance = true;
        if (!checkGrid(param)) advance = advanceBlockDim(param);
        return advance;
      } else {
        return false;
      }
    }

    unsigned int sharedBytesPerThread() const { return 0; }
  
  public:
    DomainWallDslash4DPCCuda(cudaColorSpinorField *out, const GaugeField &gauge, const cudaColorSpinorField *in,
			     const cudaColorSpinorField *x, const double mferm,
			     const double a, const double b, const int parity, const int dagger, const int *commOverride, const int DS_type)
      : DslashCuda(out, in, x, gauge, parity, dagger, commOverride), DS_type(DS_type)
    { 
      dslashParam.a = a;
      dslashParam.a_f = a;
      dslashParam.b = b;
      dslashParam.b_f = b;
      dslashParam.mferm = mferm;
      dslashParam.mferm_f = mferm;
    }
    virtual ~DomainWallDslash4DPCCuda() { unbindSpinorTex<sFloat>(in, out, x); }

    TuneKey tuneKey() const
    {
      TuneKey key = DslashCuda::tuneKey();
      switch(DS_type){
      case 0:
	strcat(key.aux,",Dslash4");
	break;
      case 1:
	strcat(key.aux,",Dslash5");
	break;
      case 2:
	strcat(key.aux,",Dslash5inv");
	break;
      }
      return key;
    }

    virtual void initTuneParam(TuneParam &param) const
    {
      Tunable::initTuneParam(param);
      param.grid = dim3( (dslashParam.threads+param.block.x-1) / param.block.x, 
			 (in->X(4)+param.block.y-1) / param.block.y, 1);
      bool ok = true;
      if (!checkGrid(param)) ok = advanceBlockDim(param);
      if (!ok) errorQuda("Lattice volume is too large for even the largest blockDim");
    }

    /** sets default values for when tuning is disabled */
    virtual void defaultTuneParam(TuneParam &param) const
    {
      Tunable::defaultTuneParam(param);
      param.grid = dim3( (dslashParam.threads+param.block.x-1) / param.block.x, 
			 (in->X(4)+param.block.y-1) / param.block.y, 1);
      bool ok = true;
      if (!checkGrid(param)) ok = advanceBlockDim(param);
      if (!ok) errorQuda("Lattice volume is too large for even the largest blockDim");
    }

    void apply(const cudaStream_t &stream)
    {
#ifndef USE_TEXTURE_OBJECTS
      if (dslashParam.kernel_type == INTERIOR_KERNEL) bindSpinorTex<sFloat>(in, out, x);
#endif // USE_TEXTURE_OBJECTS
      TuneParam tp = tuneLaunch(*this, getTuning(), getVerbosity());
      setParam();
      
      switch(DS_type){
        case 0:
          DSLASH(domainWallDslash4, tp.grid, tp.block, tp.shared_bytes, stream, dslashParam);
          break;
        case 1:
          DSLASH(domainWallDslash5, tp.grid, tp.block, tp.shared_bytes, stream, dslashParam);
          break;
        case 2:
          DSLASH(domainWallDslash5inv, tp.grid, tp.block, tp.shared_bytes, stream, dslashParam);
          break;
        default:
          errorQuda("invalid Dslash type");
      }
    }

    long long flops() const {
      long long Ls = in->X(4);
      long long vol4d = in->VolumeCB() / Ls;
      long long bulk = (Ls-2)*vol4d;
      long long wall = 2*vol4d;
      long long flops = 0;
      switch(DS_type){
        case 0:
          flops = DslashCuda::flops();
          break;
        case 1:
          flops = (x ? 48ll : 0 ) * in->VolumeCB() + 96ll*bulk + 120ll*wall;
          break;
        case 2:
          flops = 144ll*in->VolumeCB()*Ls + 3ll*Ls*(Ls-1ll);
          break;
        default:
          errorQuda("invalid Dslash type");
      }
      return flops;
    }

    long long bytes() const {
      bool isFixed = (in->Precision() == sizeof(short) || in->Precision() == sizeof(char)) ? true : false;
      int spinor_bytes = 2 * in->Ncolor() * in->Nspin() * in->Precision() + (isFixed ? sizeof(float) : 0);
      long long Ls = in->X(4);
      long long bytes = 0;

      switch(DS_type){
      case 0:
	bytes = DslashCuda::bytes();
	break;
      case 1:
	bytes = (x ? 5ll : 4ll ) * spinor_bytes * in->VolumeCB();
	break;
      case 2:
	bytes = (x ? Ls + 2 : Ls + 1) * spinor_bytes * in->VolumeCB();
	break;
      default:
	errorQuda("invalid Dslash type");
      }
      return bytes;
    }
  };
#endif // GPU_DOMAIN_WALL_DIRAC

  //-----------------------------------------------------
  // Modification for 4D preconditioned DWF operator
  // Additional Arg. is added to give a function name.
  //
  // pre-defined DS_type list
  // 0 = dslash4
  // 1 = dslash5
  // 2 = dslash5inv
  //-----------------------------------------------------

  void domainWallDslashCuda(cudaColorSpinorField *out, const cudaGaugeField &gauge, 
			    const cudaColorSpinorField *in, const int parity, const int dagger, 
			    const cudaColorSpinorField *x, const double &m_f, const double &a, const double &b,
			    const int *commOverride, const int DS_type, TimeProfile &profile)
  {
#ifdef GPU_DOMAIN_WALL_DIRAC
    using namespace dslash;
    const_cast<cudaColorSpinorField*>(in)->createComms(1);

    DslashCuda *dslash = nullptr;
    if (in->Precision() == QUDA_DOUBLE_PRECISION) {
      dslash = new DomainWallDslash4DPCCuda<double2,double2>(out, gauge, in, x, m_f, a, b, parity, dagger, commOverride, DS_type);
    } else if (in->Precision() == QUDA_SINGLE_PRECISION) {
      dslash = new DomainWallDslash4DPCCuda<float4,float4>(out, gauge, in, x, m_f, a, b, parity, dagger, commOverride, DS_type);
    } else if (in->Precision() == QUDA_HALF_PRECISION) {
      dslash = new DomainWallDslash4DPCCuda<short4,short4>(out, gauge, in, x, m_f, a, b, parity, dagger, commOverride, DS_type);
    }

    // the parameters passed to dslashCuda must be 4-d volume and 3-d
    // faces because Ls is added as the y-dimension in thread space
    int ghostFace[QUDA_MAX_DIM];
    for (int i=0; i<4; i++) ghostFace[i] = in->GhostFace()[i] / in->X(4);

    DslashPolicyImp<DslashCuda> *dslashImp = nullptr;
    if (DS_type != 0) {
      dslashImp = DslashFactory<DslashCuda>::create(QudaDslashPolicy::QUDA_DSLASH_NC);
      (*dslashImp)(*dslash, const_cast<cudaColorSpinorField*>(in), in->Volume()/in->X(4), ghostFace, profile);
      delete dslashImp;
    } else {
      DslashPolicyTune<DslashCuda> dslash_policy(
          *dslash, const_cast<cudaColorSpinorField *>(in), in->Volume() / in->X(4), ghostFace, profile);
      dslash_policy.apply(0);
    }

    delete dslash;
#else
    errorQuda("4D preconditioned Domain wall dslash has not been built");
#endif
  }

}

#endif
