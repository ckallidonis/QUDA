#include "hip/hip_runtime.h"
#ifndef USE_LEGACY_DSLASH

#include <dslash.h>
#include <worker.h>
#include <dslash_helper.cuh>
#include <color_spinor_field_order.h>
#include <gauge_field_order.h>
#include <color_spinor.h>
#include <dslash_helper.cuh>
#include <index_helper.cuh>
#include <gauge_field.h>
#include <uint_to_char.h>

#include <dslash_policy.cuh>
#include <kernels/laplace.cuh>

/**
   This is the laplacian derivative based on the basic gauged differential operator
*/

namespace quda
{

  /**
     @brief This is a helper class that is used to instantiate the
     correct templated kernel for the dslash.
  */
  template <typename Float, int nDim, int nColor, int nParity, bool dagger, bool xpay, KernelType kernel_type, typename Arg>
  struct LaplaceLaunch {

    // kernel name for jit compilation
    static constexpr const char *kernel = "quda::laplaceGPU";

    template <typename Dslash>
    inline static void launch(Dslash &dslash, TuneParam &tp, Arg &arg, const hipStream_t &stream)
    {
      dslash.launch(laplaceGPU<Float, nDim, nColor, nParity, dagger, xpay, kernel_type, Arg>, tp, arg, stream);
    }
  };

  template <typename Float, int nDim, int nColor, typename Arg> class Laplace : public Dslash<Float>
  {

protected:
    Arg &arg;
    const ColorSpinorField &in;

public:
    Laplace(Arg &arg, const ColorSpinorField &out, const ColorSpinorField &in) :
      Dslash<Float>(arg, out, in, "kernels/laplace.cuh"),
      arg(arg),
      in(in)
    {
    }

    virtual ~Laplace() {}

    void apply(const hipStream_t &stream)
    {
      TuneParam tp = tuneLaunch(*this, getTuning(), getVerbosity());
      Dslash<Float>::setParam(arg);
      Dslash<Float>::template instantiate<LaplaceLaunch, nDim, nColor>(tp, arg, stream);
    }

    long long flops() const
    {
      int mv_flops = (8 * in.Ncolor() - 2) * in.Ncolor(); // SU(3) matrix-vector flops
      int num_mv_multiply = in.Nspin() == 4 ? 2 : 1;
      int ghost_flops = (num_mv_multiply * mv_flops + 2 * in.Ncolor() * in.Nspin());
      int xpay_flops = 2 * 2 * in.Ncolor() * in.Nspin(); // multiply and add per real component
      int num_dir = (arg.dir == 4 ? 2 * 4 : 2 * 3);      // 3D or 4D operator

      long long flops_ = 0;

      // FIXME - should we count the xpay flops in the derived kernels
      // since some kernels require the xpay in the exterior (preconditiond clover)

      switch (arg.kernel_type) {
      case EXTERIOR_KERNEL_X:
      case EXTERIOR_KERNEL_Y:
      case EXTERIOR_KERNEL_Z:
      case EXTERIOR_KERNEL_T:
        flops_ = (ghost_flops + (arg.xpay ? xpay_flops : xpay_flops / 2)) * 2 * in.GhostFace()[arg.kernel_type];
        break;
      case EXTERIOR_KERNEL_ALL: {
        long long ghost_sites = 2 * (in.GhostFace()[0] + in.GhostFace()[1] + in.GhostFace()[2] + in.GhostFace()[3]);
        flops_ = (ghost_flops + (arg.xpay ? xpay_flops : xpay_flops / 2)) * ghost_sites;
        break;
      }
      case INTERIOR_KERNEL:
      case KERNEL_POLICY: {
        long long sites = in.Volume();
        flops_ = (num_dir * (in.Nspin() / 4) * in.Ncolor() * in.Nspin() + // spin project (=0 for staggered)
                  num_dir * num_mv_multiply * mv_flops +                  // SU(3) matrix-vector multiplies
                  ((num_dir - 1) * 2 * in.Ncolor() * in.Nspin()))
          * sites; // accumulation
        if (arg.xpay) flops_ += xpay_flops * sites;

        if (arg.kernel_type == KERNEL_POLICY) break;
        // now correct for flops done by exterior kernel
        long long ghost_sites = 0;
        for (int d = 0; d < 4; d++)
          if (arg.commDim[d]) ghost_sites += 2 * in.GhostFace()[d];
        flops_ -= ghost_flops * ghost_sites;

        break;
      }
      }

      return flops_;
    }

    virtual long long bytes() const
    {
      int gauge_bytes = arg.reconstruct * in.Precision();
      bool isFixed = (in.Precision() == sizeof(short) || in.Precision() == sizeof(char)) ? true : false;
      int spinor_bytes = 2 * in.Ncolor() * in.Nspin() * in.Precision() + (isFixed ? sizeof(float) : 0);
      int proj_spinor_bytes = in.Nspin() == 4 ? spinor_bytes / 2 : spinor_bytes;
      int ghost_bytes = (proj_spinor_bytes + gauge_bytes) + 2 * spinor_bytes; // 2 since we have to load the partial
      int num_dir = (arg.dir == 4 ? 2 * 4 : 2 * 3);                           // 3D or 4D operator

      long long bytes_ = 0;

      switch (arg.kernel_type) {
      case EXTERIOR_KERNEL_X:
      case EXTERIOR_KERNEL_Y:
      case EXTERIOR_KERNEL_Z:
      case EXTERIOR_KERNEL_T: bytes_ = ghost_bytes * 2 * in.GhostFace()[arg.kernel_type]; break;
      case EXTERIOR_KERNEL_ALL: {
        long long ghost_sites = 2 * (in.GhostFace()[0] + in.GhostFace()[1] + in.GhostFace()[2] + in.GhostFace()[3]);
        bytes_ = ghost_bytes * ghost_sites;
        break;
      }
      case INTERIOR_KERNEL:
      case KERNEL_POLICY: {
        long long sites = in.Volume();
        bytes_ = (num_dir * gauge_bytes + ((num_dir - 2) * spinor_bytes + 2 * proj_spinor_bytes) + spinor_bytes) * sites;
        if (arg.xpay) bytes_ += spinor_bytes;
	
        if (arg.kernel_type == KERNEL_POLICY) break;
        // now correct for bytes done by exterior kernel
        long long ghost_sites = 0;
        for (int d = 0; d < 4; d++)
          if (arg.commDim[d]) ghost_sites += 2 * in.GhostFace()[d];
        bytes_ -= ghost_bytes * ghost_sites;
	
        break;
      }
      }
      return bytes_;
    }
    
    TuneKey tuneKey() const
    {
      // add laplace transverse dir to the key
      char aux[TuneKey::aux_n];
      strcpy(aux, Dslash<Float>::aux[arg.kernel_type]);
      strcat(aux, ",laplace3D=");
      char laplace3D[32];
      u32toa(laplace3D, arg.dir);
      strcat(aux, laplace3D);
      return TuneKey(in.VolString(), typeid(*this).name(), aux);
    }
  };

  template <typename Float, int nColor, QudaReconstructType recon> struct LaplaceApply {

    inline LaplaceApply(ColorSpinorField &out, const ColorSpinorField &in, const GaugeField &U, int dir, double a,
                        const ColorSpinorField &x, int parity, bool dagger, const int *comm_override,
                        TimeProfile &profile)
    {

      constexpr int nDim = 4;
      LaplaceArg<Float, nColor, recon> arg(out, in, U, dir, a, x, parity, dagger, comm_override);
      Laplace<Float, nDim, nColor, LaplaceArg<Float, nColor, recon>> laplace(arg, out, in);

      dslash::DslashPolicyTune<decltype(laplace)> policy(
        laplace, const_cast<cudaColorSpinorField *>(static_cast<const cudaColorSpinorField *>(&in)), in.VolumeCB(),
        in.GhostFaceCB(), profile);
      policy.apply(0);

      checkCudaError();
    }
  };

  // Apply the Laplace operator
  // out(x) = M*in = - kappa*\sum_mu U_{-\mu}(x)in(x+mu) + U^\dagger_mu(x-mu)in(x-mu)
  // Uses the kappa normalization for the Wilson operator.
  // Omits direction 'dir' from the operator.
  void ApplyLaplace(ColorSpinorField &out, const ColorSpinorField &in, const GaugeField &U, int dir, double kappa,
                    const ColorSpinorField &x, int parity, bool dagger, const int *comm_override, TimeProfile &profile)
  {

    if (in.V() == out.V()) errorQuda("Aliasing pointers");
    if (in.FieldOrder() != out.FieldOrder())
      errorQuda("Field order mismatch in = %d, out = %d", in.FieldOrder(), out.FieldOrder());

    // check all precisions match
    checkPrecision(out, in, U);

    // check all locations match
    checkLocation(out, in, U);

    instantiate<LaplaceApply>(out, in, U, dir, kappa, x, parity, dagger, comm_override, profile);
  }
} // namespace quda

#else

#include <gauge_field.h>
#include <gauge_field_order.h>
#include <color_spinor_field.h>
#include <color_spinor_field_order.h>
#include <index_helper.cuh>
#include <dslash_quda.h>
#include <color_spinor.h>
#include <worker.h>
#include <tune_quda.h>

/**
   This is a basic gauged Laplace operator
*/

namespace quda {

  /**
     @brief Parameter structure for driving the Laplace operator
   */
  template <typename Float, int nColor, QudaReconstructType reconstruct, bool xpay>
  struct LaplaceArg {
    typedef typename colorspinor_mapper<Float,1,nColor>::type F;
    typedef typename gauge_mapper<Float,reconstruct>::type G;

    F out;                // output vector field
    const F in;           // input vector field
    const F x;            // input vector when doing xpay
    const G U;            // the gauge field
    const Float kappa;    // kappa parameter = 1/(8+m)
    const int parity;     // only use this for single parity fields
    const int nParity;    // number of parities we're working on
    const int nFace;      // hard code to 1 for now
    const int dim[5];     // full lattice dimensions
    const int commDim[4]; // whether a given dimension is partitioned or not
    const int volumeCB;   // checkerboarded volume

    __host__ __device__ static constexpr bool isXpay() { return xpay; }

    LaplaceArg(ColorSpinorField &out, const ColorSpinorField &in, const GaugeField &U,
	       Float kappa, const ColorSpinorField *x, int parity)
      : out(out), in(in), U(U), kappa(kappa), x(xpay ? *x : in), parity(parity), nParity(in.SiteSubset()), nFace(1),
	dim{ (3-nParity) * in.X(0), in.X(1), in.X(2), in.X(3), 1 },
      commDim{comm_dim_partitioned(0), comm_dim_partitioned(1), comm_dim_partitioned(2), comm_dim_partitioned(3)},
      volumeCB(in.VolumeCB())
    {
      if (in.FieldOrder() != QUDA_FLOAT2_FIELD_ORDER || !U.isNative())
      errorQuda("Unsupported field order colorspinor=%d gauge=%d combination\n", in.FieldOrder(), U.FieldOrder());
    }
  };

  /**
     Applies the off-diagonal part of the Laplace operator

     @param[out] out The out result field
     @param[in] U The gauge field
     @param[in] kappa Kappa value
     @param[in] in The input field
     @param[in] parity The site parity
     @param[in] x_cb The checkerboarded site index
   */
  template <typename Float, int nDim, int nColor, typename Vector, typename Arg>
  __device__ __host__ inline void applyLaplace(Vector &out, Arg &arg, int x_cb, int parity) {
    typedef Matrix<complex<Float>,nColor> Link;
    const int their_spinor_parity = (arg.nParity == 2) ? 1-parity : 0;

    int coord[5];
    getCoords(coord, x_cb, arg.dim, parity);
    coord[4] = 0;

#pragma unroll
    for (int d = 0; d<nDim; d++) // loop over dimension
    {
      //Forward gather - compute fwd offset for vector fetch
      const int fwd_idx = linkIndexP1(coord, arg.dim, d);

      if ( arg.commDim[d] && (coord[d] + arg.nFace >= arg.dim[d]) ) {
	const int ghost_idx = ghostFaceIndex<1>(coord, arg.dim, d, arg.nFace);

	const Link U = arg.U(d, x_cb, parity);
	const Vector in = arg.in.Ghost(d, 1, ghost_idx, their_spinor_parity);

	out += U * in;
      } else {

        const Link U = arg.U(d, x_cb, parity);
	const Vector in = arg.in(fwd_idx, their_spinor_parity);

	out += U * in;
      }

      //Backward gather - compute back offset for spinor and gauge fetch
      const int back_idx = linkIndexM1(coord, arg.dim, d);
      const int gauge_idx = back_idx;

      if ( arg.commDim[d] && (coord[d] - arg.nFace < 0) ) {
	const int ghost_idx = ghostFaceIndex<0>(coord, arg.dim, d, arg.nFace);

	const Link U = arg.U.Ghost(d, ghost_idx, 1-parity);
	const Vector in = arg.in.Ghost(d, 0, ghost_idx, their_spinor_parity);

	out += conj(U) * in;
      } else {
	
	const Link U = arg.U(d, gauge_idx, 1-parity);
	const Vector in = arg.in(back_idx, their_spinor_parity);

	out += conj(U) * in;
      }
    } //nDim
  }


  //out(x) = M*in = (-D + m) * in(x-mu)
  template <typename Float, int nDim, int nColor, typename Arg>
  __device__ __host__ inline void laplace(Arg &arg, int x_cb, int parity)
  {
    typedef ColorSpinor<Float,nColor,1> Vector;
    Vector out;

    applyLaplace<Float,nDim,nColor>(out, arg, x_cb, parity);

    if (arg.isXpay()) {
      Vector x = arg.x(x_cb, parity);
      out = x + arg.kappa * out;
    }
    arg.out(x_cb, arg.nParity == 2 ? parity : 0) = out;
  }

  // CPU kernel for applying the Laplace operator to a vector
  template <typename Float, int nDim, int nColor, typename Arg>
  void laplaceCPU(Arg arg)
  {

    for (int parity= 0; parity < arg.nParity; parity++) {
      // for full fields then set parity from loop else use arg setting
      parity = (arg.nParity == 2) ? parity : arg.parity;

      for (int x_cb = 0; x_cb < arg.volumeCB; x_cb++) { // 4-d volume
	laplace<Float,nDim,nColor>(arg, x_cb, parity);
      } // 4-d volumeCB
    } // parity

  }

  // GPU Kernel for applying the Laplace operator to a vector
  template <typename Float, int nDim, int nColor, typename Arg>
  __global__ void laplaceGPU(Arg arg)
  {
    int x_cb = blockIdx.x*blockDim.x + threadIdx.x;

    // for full fields set parity from y thread index else use arg setting
    int parity = (arg.nParity == 2) ? blockDim.y*blockIdx.y + threadIdx.y : arg.parity;

    if (x_cb >= arg.volumeCB) return;
    if (parity >= arg.nParity) return;

    laplace<Float,nDim,nColor>(arg, x_cb, parity);
  }

  template <typename Float, int nDim, int nColor, typename Arg>
  class Laplace : public TunableVectorY {

  protected:
    Arg &arg;
    const ColorSpinorField &meta;

    long long flops() const
    {
      return (2*nDim*(8*nColor*nColor)-2*nColor + (arg.isXpay() ? 2*2*nColor : 0) )*arg.nParity*(long long)meta.VolumeCB();
    }
    long long bytes() const
    {
      return arg.out.Bytes() + 2*nDim*arg.in.Bytes() + arg.nParity*2*nDim*arg.U.Bytes()*meta.VolumeCB() +
	(arg.isXpay() ? arg.x.Bytes() : 0);
    }
    bool tuneGridDim() const { return false; }
    unsigned int minThreads() const { return arg.volumeCB; }

  public:
    Laplace(Arg &arg, const ColorSpinorField &meta) : TunableVectorY(arg.nParity), arg(arg), meta(meta)
    {
      strcpy(aux, meta.AuxString());
#ifdef MULTI_GPU
      char comm[5];
      comm[0] = (arg.commDim[0] ? '1' : '0');
      comm[1] = (arg.commDim[1] ? '1' : '0');
      comm[2] = (arg.commDim[2] ? '1' : '0');
      comm[3] = (arg.commDim[3] ? '1' : '0');
      comm[4] = '\0';
      strcat(aux,",comm=");
      strcat(aux,comm);
#endif
      if (arg.isXpay()) strcat(aux,",xpay");
    }
    virtual ~Laplace() { }

    void apply(const hipStream_t &stream) {
      if (meta.Location() == QUDA_CPU_FIELD_LOCATION) {
	laplaceCPU<Float,nDim,nColor>(arg);
      } else {
        TuneParam tp = tuneLaunch(*this, getTuning(), getVerbosity());
	laplaceGPU<Float,nDim,nColor> <<<tp.grid,tp.block,tp.shared_bytes,stream>>>(arg);
      }
    }

    TuneKey tuneKey() const { return TuneKey(meta.VolString(), typeid(*this).name(), aux); }
  };


  template <typename Float, int nColor, QudaReconstructType recon>
    void ApplyLaplace(ColorSpinorField &out, const ColorSpinorField &in, const GaugeField &U,
		      double kappa, const ColorSpinorField *x, int parity)
  {
    constexpr int nDim = 4;
    if (x) {
      LaplaceArg<Float,nColor,recon,true> arg(out, in, U, kappa, x, parity);
      Laplace<Float,nDim,nColor,LaplaceArg<Float,nColor,recon,true> > laplace(arg, in);
      laplace.apply(0);
    } else {
      LaplaceArg<Float,nColor,recon,false> arg(out, in, U, kappa, x, parity);
      Laplace<Float,nDim,nColor,LaplaceArg<Float,nColor,recon,false> > laplace(arg, in);
      laplace.apply(0);
    }
  }

  // template on the gauge reconstruction
  template <typename Float, int nColor>
    void ApplyLaplace(ColorSpinorField &out, const ColorSpinorField &in, const GaugeField &U,
		      double kappa, const ColorSpinorField *x, int parity)
  {
    if (U.Reconstruct()== QUDA_RECONSTRUCT_NO) {
      ApplyLaplace<Float,nColor,QUDA_RECONSTRUCT_NO>(out, in, U, kappa, x, parity);
    } else if (U.Reconstruct()== QUDA_RECONSTRUCT_12) {
      ApplyLaplace<Float,nColor,QUDA_RECONSTRUCT_12>(out, in, U, kappa, x, parity);
    } else if (U.Reconstruct()== QUDA_RECONSTRUCT_8) {
      ApplyLaplace<Float,nColor,QUDA_RECONSTRUCT_8>(out, in, U, kappa, x, parity);
    } else {
      errorQuda("Unsupported reconstruct type %d\n", U.Reconstruct());
    }
  }

  // template on the number of colors
  template <typename Float>
    void ApplyLaplace(ColorSpinorField &out, const ColorSpinorField &in, const GaugeField &U,
		      double kappa, const ColorSpinorField *x, int parity)
  {
    if (in.Ncolor() == 3) {
      ApplyLaplace<Float,3>(out, in, U, kappa, x, parity);
    } else {
      errorQuda("Unsupported number of colors %d\n", U.Ncolor());
    }
  }

  // this is the Worker pointer that may have issue additional work
  // while we're waiting on communication to finish
  namespace dslash {
    extern Worker* aux_worker;
  }

  //Apply the Laplace operator
  //out(x) = M*in = - kappa*\sum_mu U_{-\mu}(x)in(x+mu) + U^\dagger_mu(x-mu)in(x-mu)
  //Uses the kappa normalization for the Wilson operator.
  void ApplyLaplace(ColorSpinorField &out, const ColorSpinorField &in, const GaugeField &U,
		    double kappa, const ColorSpinorField *x, int parity)		    
  {
    if (in.V() == out.V()) errorQuda("Aliasing pointers");
    if (in.FieldOrder() != out.FieldOrder())
      errorQuda("Field order mismatch in = %d, out = %d", in.FieldOrder(), out.FieldOrder());
    
    // check all precisions match
    checkPrecision(out, in, U);

    // check all locations match
    checkLocation(out, in, U);

    const int nFace = 1;
    in.exchangeGhost((QudaParity)(1-parity), nFace, 0); // last parameter is dummy

    if (dslash::aux_worker) dslash::aux_worker->apply(0);

    if (U.Precision() == QUDA_DOUBLE_PRECISION) {
      ApplyLaplace<double>(out, in, U, kappa, x, parity);
    } else if (U.Precision() == QUDA_SINGLE_PRECISION) {
      ApplyLaplace<float>(out, in, U, kappa, x, parity);
    } else {
      errorQuda("Unsupported precision %d\n", U.Precision());
    }

    in.bufferIndex = (1 - in.bufferIndex);
  }
} // namespace quda

#endif
