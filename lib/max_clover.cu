#include "hip/hip_runtime.h"
#include <clover_field_order.h>

namespace quda {

  using namespace clover;

  enum norm_type_ {
    NORM1,
    NORM2,
    ABS_MAX,
    ABS_MIN
  };

  template<typename real, int Nc, QudaCloverFieldOrder order>
  double norm(const CloverField &u, norm_type_ type) {
    constexpr int Ns = 4;
    typedef typename mapper<real>::type reg_type;
    double norm_ = 0.0;
    switch(type) {
    case   NORM1: norm_ = FieldOrder<reg_type,Nc,Ns,order>(const_cast<CloverField &>(u)).norm1();   break;
    case   NORM2: norm_ = FieldOrder<reg_type,Nc,Ns,order>(const_cast<CloverField &>(u)).norm2();   break;
    case ABS_MAX: norm_ = FieldOrder<reg_type,Nc,Ns,order>(const_cast<CloverField &>(u)).abs_max(); break;
    case ABS_MIN: norm_ = FieldOrder<reg_type,Nc,Ns,order>(const_cast<CloverField &>(u)).abs_min(); break;
    }
    return norm_;
  }

  template<typename real, int Nc>
  double norm(const CloverField &u, norm_type_ type) {
    double norm_ = 0.0;
    switch (u.Order()) {
    case QUDA_FLOAT2_CLOVER_ORDER: norm_ = norm<real,Nc,QUDA_FLOAT2_CLOVER_ORDER>(u, type); break;
    case QUDA_FLOAT4_CLOVER_ORDER: norm_ = norm<real,Nc,QUDA_FLOAT4_CLOVER_ORDER>(u, type); break;
    default: errorQuda("Clover field %d order not supported", u.Order());
    }
    return norm_;
  }

  template<typename real>
  double _norm(const CloverField &u, norm_type_ type) {
    double norm_ = 0.0;
    switch(u.Ncolor()) {
    case  3: norm_ = norm<real, 3>(u, type); break;
    default: errorQuda("Unsupported color %d", u.Ncolor());
    }
    return norm_;
  }

  double CloverField::norm1() const {
    double nrm1 = 0.0;
    switch(precision) {
    case QUDA_DOUBLE_PRECISION: nrm1 = _norm<double>(*this, NORM1); break;
    case QUDA_SINGLE_PRECISION: nrm1 = _norm< float>(*this, NORM1); break;
    default: errorQuda("Unsupported precision %d", precision);
    }
    return nrm1;
  }

  double CloverField::norm2() const {
    double nrm2 = 0.0;
    switch(precision) {
    case QUDA_DOUBLE_PRECISION: nrm2 = _norm<double>(*this, NORM2); break;
    case QUDA_SINGLE_PRECISION: nrm2 = _norm< float>(*this, NORM2); break;
    default: errorQuda("Unsupported precision %d", precision);
    }
    return nrm2;
  }

  double CloverField::abs_max() const {
    double max = 0.0;
    switch(precision) {
    case QUDA_DOUBLE_PRECISION: max = _norm<double>(*this, ABS_MAX); break;
    case QUDA_SINGLE_PRECISION: max = _norm< float>(*this, ABS_MAX); break;
    default: errorQuda("Unsupported precision %d", precision);
    }
    return max;
  }

  double CloverField::abs_min() const {
    double min = 0.0;
    switch(precision) {
    case QUDA_DOUBLE_PRECISION: min = _norm<double>(*this, ABS_MIN); break;
    case QUDA_SINGLE_PRECISION: min = _norm< float>(*this, ABS_MIN); break;
    default: errorQuda("Unsupported precision %d", precision);
    }
    return min;
  }

} // namespace quda
