
#include <hip/hip_runtime.h>
#ifdef USE_LEGACY_DSLASH

#include <cstdlib>
#include <cstdio>
#include <string>
#include <iostream>
#include <typeinfo>

#include <color_spinor_field.h>

// these control the Wilson-type actions
#ifdef GPU_WILSON_DIRAC
//#define DIRECT_ACCESS_WILSON_PACK_SPINOR
#endif // GPU_WILSON_DIRAC


#include <quda_internal.h>
#include <dslash_quda.h>
#include <dslash.h>
#include <sys/time.h>
#include <blas_quda.h>

#include <inline_ptx.h>
#include <uint_to_char.h>

#include <index_helper.cuh>

namespace quda
{

#ifdef MULTI_GPU
  static int commDim[QUDA_MAX_DIM]; // Whether to do comms or not
  void setPackComms(const int *comm_dim) {
    for (int i=0; i<4; i++) commDim[i] = comm_dim[i];
    for (int i=4; i<QUDA_MAX_DIM; i++) commDim[i] = 0;
  }
#else
  void setPackComms(const int *comm_dim) { ; }
#endif

#define STRIPED
#ifdef STRIPED
#else
#define SWIZZLE
#endif

  namespace pack
  {

#include <dslash_constants.h>
#include <dslash_textures.h>

  } // end namespace pack

  using namespace pack;

#include <dslash_index.cuh>

  // routines for packing the ghost zones (multi-GPU only)


#ifdef MULTI_GPU

  template <typename FloatN>
  struct PackParam {

    FloatN *out[2*4];
    float *outNorm[2*4];

    FloatN *in;
    float *inNorm;

    int_fastdiv threads; // total number of threads

    // offsets which determine thread mapping to dimension
    int threadDimMapLower[4]; // lowest thread which maps to dim
    int threadDimMapUpper[4]; // greatest thread + 1 which maps to dim

    int parity;
#ifdef USE_TEXTURE_OBJECTS
    cudaTextureObject_t inTex;
    cudaTextureObject_t inTexNorm;
#endif

    int dim;
    int face_num;

    DslashConstant dc;

    int sp_stride;

    int_fastdiv swizzle;
    int sites_per_block;
  };

  template<typename FloatN>
  std::ostream& operator<<(std::ostream& output, const PackParam<FloatN>& param) {
    output << "threads = " << param.threads << std::endl;
    output << "threadDimMapLower = {" << param.threadDimMapLower[0] << "," <<
      param.threadDimMapLower[1] << "," << param.threadDimMapLower[2] << "," << param.threadDimMapLower[3] << "}" << std::endl;
    output << "threadDimMapUpper = {" << param.threadDimMapUpper[0] << "," <<
      param.threadDimMapUpper[1] << "," << param.threadDimMapUpper[2] << "," << param.threadDimMapUpper[3] << "}" << std::endl;
    output << "parity = " << param.parity << std::endl;
    output << "dim = " << param.dim << std::endl;
    output << "face_num = " << param.face_num << std::endl;
    output << "X = {" << param.dc.X[0] << ","<< param.dc.X[1] << "," << param.dc.X[2] << "," << param.dc.X[3] << "," << param.dc.X[4] << "}" << std::endl;
    output << "ghostFace = {" << param.dc.ghostFaceCB[0] << "," << param.dc.ghostFaceCB[1] << ","
           << param.dc.ghostFaceCB[2] << "," << param.dc.ghostFaceCB[3] << "}" << std::endl;
    output << "sp_stride = " << param.sp_stride << std::endl;

    output << "swizzle = " << param.swizzle << std::endl;
    output << "sites_per_block = " << param.sites_per_block << std::endl;
    return output;
  }

  // Extend the PackParam class to PackExtendedParam
  template<typename Float>
    struct PackExtendedParam : public PackParam<Float>
    {
      PackExtendedParam(){}
    PackExtendedParam(const PackParam<Float>& base) : PackParam<Float>(base) {}
      int R[QUDA_MAX_DIM]; // boundary dimensions
    };

#if defined(GPU_WILSON_DIRAC) || defined(GPU_DOMAIN_WALL_DIRAC)

  // double precision
#if (defined DIRECT_ACCESS_WILSON_PACK_SPINOR) || (defined FERMI_NO_DBLE_TEX)
#define READ_SPINOR READ_SPINOR_DOUBLE
#define READ_SPINOR_UP READ_SPINOR_DOUBLE_UP
#define READ_SPINOR_DOWN READ_SPINOR_DOUBLE_DOWN
#define SPINORTEX in
#else
#define READ_SPINOR READ_SPINOR_DOUBLE_TEX
#define READ_SPINOR_UP READ_SPINOR_DOUBLE_UP_TEX
#define READ_SPINOR_DOWN READ_SPINOR_DOUBLE_DOWN_TEX
#ifdef USE_TEXTURE_OBJECTS
#define SPINORTEX param.inTex
#else
#define SPINORTEX spinorTexDouble
#endif
#endif
#define WRITE_HALF_SPINOR WRITE_HALF_SPINOR_DOUBLE2
#define SPINOR_DOUBLE
  template <int dim, int dagger, int face_num>
    static inline __device__ void packFaceWilsonCore(double2 *out, float *outNorm, const double2 *in,
						     const float *inNorm, const int &idx,
						     const int &face_idx, const int &face_volume,
						     PackParam<double2> &param)
  {
    if (dagger) {
#include "wilson_pack_face_dagger_core.h"
    } else {
#include "wilson_pack_face_core.h"
    }
  }

  template <int dim, int dagger, int face_num>
    static inline __device__ void unpackFaceWilsonCore(double2 *out, float *outNorm, const double2 *in,
						       const float *inNorm, const int &idx,
						       const int &face_idx, const int &face_volume,
						       PackParam<double2> &param)
  {
    if (dagger) {
#include "wilson_pack_face_dagger_core.h"
    } else {
#include "wilson_pack_face_core.h"
    }
  }

#undef READ_SPINOR
#undef READ_SPINOR
#undef READ_SPINOR_UP
#undef READ_SPINOR_DOWN
#undef SPINORTEX
#undef WRITE_HALF_SPINOR
#undef SPINOR_DOUBLE


  // single precision
#ifdef DIRECT_ACCESS_WILSON_PACK_SPINOR
#define READ_SPINOR READ_SPINOR_SINGLE
#define READ_SPINOR_UP READ_SPINOR_SINGLE_UP
#define READ_SPINOR_DOWN READ_SPINOR_SINGLE_DOWN
#define SPINORTEX in
#else
#define READ_SPINOR READ_SPINOR_SINGLE_TEX
#define READ_SPINOR_UP READ_SPINOR_SINGLE_UP_TEX
#define READ_SPINOR_DOWN READ_SPINOR_SINGLE_DOWN_TEX
#ifdef USE_TEXTURE_OBJECTS
#define SPINORTEX param.inTex
#else
#define SPINORTEX spinorTexSingle
#endif
#endif
#define WRITE_HALF_SPINOR WRITE_HALF_SPINOR_FLOAT4
  template <int dim, int dagger, int face_num>
    static inline __device__ void packFaceWilsonCore(float4 *out, float *outNorm, const float4 *in, const float *inNorm,
						     const int &idx, const int &face_idx,
						     const int &face_volume,
						     const PackParam<float4> &param)
  {
    if (dagger) {
#include "wilson_pack_face_dagger_core.h"
    } else {
#include "wilson_pack_face_core.h"
    }
  }

  template <int dim, int dagger, int face_num>
    static inline __device__ void unpackFaceWilsonCore(float4 *out, float *outNorm, const float4 *in, const float *inNorm,
						       const int &idx, const int &face_idx,
						       const int &face_volume,
						       const PackParam<float4> &param)
  {
    if (dagger) {
#include "wilson_pack_face_dagger_core.h"
    } else {
#include "wilson_pack_face_core.h"
    }
  }
#undef READ_SPINOR
#undef READ_SPINOR_UP
#undef READ_SPINOR_DOWN
#undef SPINORTEX
#undef WRITE_HALF_SPINOR


  // half precision
#ifdef DIRECT_ACCESS_WILSON_PACK_SPINOR
#define READ_SPINOR READ_SPINOR_HALF
#define READ_SPINOR_UP READ_SPINOR_HALF_UP
#define READ_SPINOR_DOWN READ_SPINOR_HALF_DOWN
#define SPINORTEX in
#else
#define READ_SPINOR READ_SPINOR_HALF_TEX
#define READ_SPINOR_UP READ_SPINOR_HALF_UP_TEX
#define READ_SPINOR_DOWN READ_SPINOR_HALF_DOWN_TEX
#ifdef USE_TEXTURE_OBJECTS
#define SPINORTEX param.inTex
#else
#define SPINORTEX spinorTexHalf
#endif
#endif
#define WRITE_HALF_SPINOR WRITE_HALF_SPINOR_SHORT4
  template <int dim, int dagger, int face_num>
    static inline __device__ void packFaceWilsonCore(short4 *out, float *outNorm, const short4 *in, const float *inNorm,
						     const int &idx, const int &face_idx,
						     const int &face_volume,
						     const PackParam<short4> &param)
  {
    if (dagger) {
#include "wilson_pack_face_dagger_core.h"
    } else {
#include "wilson_pack_face_core.h"
    }
  }

  template <int dim, int dagger, int face_num>
    static inline __device__ void unpackFaceWilsonCore(short4 *out, float *outNorm, const short4 *in, const float *inNorm,
						       const int &idx, const int &face_idx,
						       const int &face_volume,
						       const PackParam<short4> &param)
  {
    if (dagger) {
#include "wilson_pack_face_dagger_core.h"
    } else {
#include "wilson_pack_face_core.h"
    }
  }
#undef READ_SPINOR
#undef READ_SPINOR_UP
#undef READ_SPINOR_DOWN
#undef SPINORTEX
#undef WRITE_HALF_SPINOR

  // quarter precision
#ifdef DIRECT_ACCESS_WILSON_PACK_SPINOR
#define READ_SPINOR READ_SPINOR_QUARTER
#define READ_SPINOR_UP READ_SPINOR_QUARTER_UP
#define READ_SPINOR_DOWN READ_SPINOR_QUARTER_DOWN
#define SPINORTEX in
#else
#define READ_SPINOR READ_SPINOR_QUARTER_TEX
#define READ_SPINOR_UP READ_SPINOR_QUARTER_UP_TEX
#define READ_SPINOR_DOWN READ_SPINOR_QUARTER_DOWN_TEX
#ifdef USE_TEXTURE_OBJECTS
#define SPINORTEX param.inTex
#else
#define SPINORTEX spinorTexHalf
#endif
#endif
#define WRITE_HALF_SPINOR WRITE_HALF_SPINOR_CHAR4
  template <int dim, int dagger, int face_num>
    static inline __device__ void packFaceWilsonCore(char4 *out, float *outNorm, const char4 *in, const float *inNorm,
                 const int &idx, const int &face_idx, 
                 const int &face_volume, 
                 const PackParam<char4> &param)
  {
    if (dagger) {
#include "wilson_pack_face_dagger_core.h"
    } else {
#include "wilson_pack_face_core.h"
    }
  }

  template <int dim, int dagger, int face_num>
    static inline __device__ void unpackFaceWilsonCore(char4 *out, float *outNorm, const char4 *in, const float *inNorm,
                   const int &idx, const int &face_idx, 
                   const int &face_volume, 
                   const PackParam<char4> &param)
  {
    if (dagger) {
#include "wilson_pack_face_dagger_core.h"
    } else {
#include "wilson_pack_face_core.h"
    }
  }
#undef READ_SPINOR
#undef READ_SPINOR_UP
#undef READ_SPINOR_DOWN
#undef SPINORTEX
#undef WRITE_HALF_SPINOR


  template <int dagger, typename FloatN>
    __global__ void packFaceWilsonKernel(PackParam<FloatN> param)
  {
    const int nFace = 1; // 1 face for Wilson

#ifdef STRIPED
    const int sites_per_block = param.sites_per_block;
    int local_tid = threadIdx.x;
    int tid = sites_per_block * blockIdx.x + local_tid;
#else
    int tid = block_idx(param.swizzle) * blockDim.x + threadIdx.x;
    constexpr int sites_per_block = 1;
    constexpr int local_tid = 0;
#endif

    while ( local_tid < sites_per_block && tid < param.threads ) {

      // determine which dimension we are packing
      int face_idx;
      const int dim = dimFromFaceIndex(face_idx, tid, param);

      // compute where the output is located
      // compute an index into the local volume from the index into the face
      // read spinor, spin-project, and write half spinor to face
      if (dim == 0) {
        // face_num determines which end of the lattice we are packing: 0 = start, 1 = end
        const int face_num = (face_idx >= nFace * param.dc.ghostFaceCB[0]) ? 1 : 0;
        face_idx -= face_num * nFace * param.dc.ghostFaceCB[0];
        if (face_num == 0) {
          const int idx = indexFromFaceIndex<4,QUDA_4D_PC,0,nFace,0>(face_idx,param);
          packFaceWilsonCore<0, dagger, 0>(
              param.out[0], param.outNorm[0], param.in, param.inNorm, idx, face_idx, param.dc.ghostFaceCB[0], param);
        } else {
          const int idx = indexFromFaceIndex<4,QUDA_4D_PC,0,nFace,1>(face_idx,param);
          packFaceWilsonCore<0, dagger, 1>(
              param.out[1], param.outNorm[1], param.in, param.inNorm, idx, face_idx, param.dc.ghostFaceCB[0], param);
        }
      } else if (dim == 1) {
        // face_num determines which end of the lattice we are packing: 0 = start, 1 = end
        const int face_num = (face_idx >= nFace * param.dc.ghostFaceCB[1]) ? 1 : 0;
        face_idx -= face_num * nFace * param.dc.ghostFaceCB[1];
        if (face_num == 0) {
          const int idx = indexFromFaceIndex<4,QUDA_4D_PC,1,nFace,0>(face_idx,param);
          packFaceWilsonCore<1, dagger, 0>(
              param.out[2], param.outNorm[2], param.in, param.inNorm, idx, face_idx, param.dc.ghostFaceCB[1], param);
        } else {
          const int idx = indexFromFaceIndex<4,QUDA_4D_PC,1,nFace,1>(face_idx,param);
          packFaceWilsonCore<1, dagger, 1>(
              param.out[3], param.outNorm[3], param.in, param.inNorm, idx, face_idx, param.dc.ghostFaceCB[1], param);
        }
      } else if (dim == 2) {
        // face_num determines which end of the lattice we are packing: 0 = start, 1 = end
        const int face_num = (face_idx >= nFace * param.dc.ghostFaceCB[2]) ? 1 : 0;
        face_idx -= face_num * nFace * param.dc.ghostFaceCB[2];
        if (face_num == 0) {
          const int idx = indexFromFaceIndex<4,QUDA_4D_PC,2,nFace,0>(face_idx,param);
          packFaceWilsonCore<2, dagger, 0>(
              param.out[4], param.outNorm[4], param.in, param.inNorm, idx, face_idx, param.dc.ghostFaceCB[2], param);
        } else {
          const int idx = indexFromFaceIndex<4,QUDA_4D_PC,2,nFace,1>(face_idx,param);
          packFaceWilsonCore<2, dagger, 1>(
              param.out[5], param.outNorm[5], param.in, param.inNorm, idx, face_idx, param.dc.ghostFaceCB[2], param);
        }
      } else {
        // face_num determines which end of the lattice we are packing: 0 = start, 1 = end
        const int face_num = (face_idx >= nFace * param.dc.ghostFaceCB[3]) ? 1 : 0;
        face_idx -= face_num * nFace * param.dc.ghostFaceCB[3];
        if (face_num == 0) {
          const int idx = indexFromFaceIndex<4,QUDA_4D_PC,3,nFace,0>(face_idx,param);
          packFaceWilsonCore<3, dagger, 0>(
              param.out[6], param.outNorm[6], param.in, param.inNorm, idx, face_idx, param.dc.ghostFaceCB[3], param);
        } else {
          const int idx = indexFromFaceIndex<4,QUDA_4D_PC,3,nFace,1>(face_idx,param);
          packFaceWilsonCore<3, dagger, 1>(
              param.out[7], param.outNorm[7], param.in, param.inNorm, idx, face_idx, param.dc.ghostFaceCB[3], param);
        }
      }

#ifdef STRIPED
      local_tid += blockDim.x;
      tid += blockDim.x;
#else
      tid += blockDim.x*gridDim.x;
#endif
    } // while tid

  }


  template <int dagger, typename FloatN, int nFace>
    __global__ void packFaceExtendedWilsonKernel(PackParam<FloatN> param)
  {

#ifdef STRIPED
    const int sites_per_block = param.sites_per_block;
    int local_tid = threadIdx.x;
    int tid = sites_per_block * blockIdx.x + local_tid;
#else
    int tid = block_idx(param.swizzle) * blockDim.x + threadIdx.x;
    constexpr int sites_per_block = 1;
    constexpr int local_tid = 0;
#endif

    while ( local_tid < sites_per_block && tid < param.threads ) {

      // determine which dimension we are packing
      int face_idx;
      const int dim = dimFromFaceIndex(face_idx, tid, param);

      // compute where the output is located
      // compute an index into the local volume from the index into the face
      // read spinor, spin-project, and write half spinor to face
      if (dim == 0) {
        // face_num determines which end of the lattice we are packing: 0 = start, 1 = end
        // if param.face_num==2 pack both the start and the end, otherwise pack the region of the lattice
        // specified by param.face_num
        const int face_num
            = (param.face_num == 2) ? ((face_idx >= nFace * param.dc.ghostFaceCB[0]) ? 1 : 0) : param.face_num;
        if (param.face_num == 2) face_idx -= face_num * nFace * param.dc.ghostFaceCB[0];
        if (face_num == 0) {
          const int idx = indexFromFaceIndexExtended<0,nFace,0>(face_idx,param);
          packFaceWilsonCore<0, dagger, 0>(
              param.out[0], param.outNorm[0], param.in, param.inNorm, idx, face_idx, param.dc.ghostFaceCB[0], param);
        } else {
          const int idx = indexFromFaceIndexExtended<0,nFace,1>(face_idx,param);
          packFaceWilsonCore<0, dagger, 1>(
              param.out[1], param.outNorm[1], param.in, param.inNorm, idx, face_idx, param.dc.ghostFaceCB[0], param);
        }
      } else if (dim == 1) {
        const int face_num
            = (param.face_num == 2) ? ((face_idx >= nFace * param.dc.ghostFaceCB[1]) ? 1 : 0) : param.face_num;
        if (param.face_num == 2) face_idx -= face_num * nFace * param.dc.ghostFaceCB[1];
        if (face_num == 0) {
          const int idx = indexFromFaceIndexExtended<1,nFace,0>(face_idx,param);
          packFaceWilsonCore<1, dagger, 0>(
              param.out[2], param.outNorm[2], param.in, param.inNorm, idx, face_idx, param.dc.ghostFaceCB[1], param);
        } else {
          const int idx = indexFromFaceIndexExtended<1,nFace,1>(face_idx,param);
          packFaceWilsonCore<1, dagger, 1>(
              param.out[3], param.outNorm[3], param.in, param.inNorm, idx, face_idx, param.dc.ghostFaceCB[1], param);
        }
      } else if (dim == 2) {
        const int face_num
            = (param.face_num == 2) ? ((face_idx >= nFace * param.dc.ghostFaceCB[2]) ? 1 : 0) : param.face_num;
        if (param.face_num == 2) face_idx -= face_num * nFace * param.dc.ghostFaceCB[2];
        if (face_num == 0) {
          const int idx = indexFromFaceIndexExtended<2,nFace,0>(face_idx,param);
          packFaceWilsonCore<2, dagger, 0>(
              param.out[4], param.outNorm[4], param.in, param.inNorm, idx, face_idx, param.dc.ghostFaceCB[2], param);
        } else {
          const int idx = indexFromFaceIndexExtended<2,nFace,1>(face_idx,param);
          packFaceWilsonCore<2, dagger, 1>(
              param.out[5], param.outNorm[5], param.in, param.inNorm, idx, face_idx, param.dc.ghostFaceCB[2], param);
        }
      } else {
        const int face_num
            = (param.face_num == 2) ? ((face_idx >= nFace * param.dc.ghostFaceCB[3]) ? 1 : 0) : param.face_num;
        if (param.face_num == 2) face_idx -= face_num * nFace * param.dc.ghostFaceCB[3];

        if (face_num == 0) {
          const int idx = indexFromFaceIndexExtended<3,nFace,0>(face_idx,param);
          packFaceWilsonCore<3, dagger, 0>(
              param.out[6], param.outNorm[6], param.in, param.inNorm, idx, face_idx, param.dc.ghostFaceCB[3], param);
        } else {
          const int idx = indexFromFaceIndexExtended<3,nFace,1>(face_idx,param);
          packFaceWilsonCore<3, dagger, 1>(
              param.out[7], param.outNorm[7], param.in, param.inNorm, idx, face_idx, param.dc.ghostFaceCB[3], param);
        }
      }

#ifdef STRIPED
      local_tid += blockDim.x;
      tid += blockDim.x;
#else
      tid += blockDim.x*gridDim.x;
#endif
    } // while tid

  }


  template <int dagger, typename FloatN, int nFace>
    __global__ void unpackFaceExtendedWilsonKernel(PackParam<FloatN> param)
  {

#ifdef STRIPED
    const int sites_per_block = param.sites_per_block;
    int local_tid = threadIdx.x;
    int tid = sites_per_block * blockIdx.x + local_tid;
#else
    int tid = block_idx(param.swizzle) * blockDim.x + threadIdx.x;
    constexpr int sites_per_block = 1;
    constexpr int local_tid = 0;
#endif

    while ( local_tid < sites_per_block && tid < param.threads ) {

      // determine which dimension we are packing
      int face_idx;
      const int dim = dimFromFaceIndex(face_idx, tid, param);

      // compute where the output is located
      // compute an index into the local volume from the index into the face
      // read spinor, spin-project, and write half spinor to face
      if (dim == 0) {
        // face_num determines which end of the lattice we are packing: 0 = start, 1 = end
        // if param.face_num==2 pack both the start and the end, otherwise pack the region of the lattice
        // specified by param.face_num
        const int face_num
            = (param.face_num == 2) ? ((face_idx >= nFace * param.dc.ghostFaceCB[0]) ? 1 : 0) : param.face_num;
        if (param.face_num == 2) face_idx -= face_num * nFace * param.dc.ghostFaceCB[0];

        if (face_num == 0) {
          const int idx = indexFromFaceIndexExtended<0,nFace,0>(face_idx,param);
          unpackFaceWilsonCore<0, dagger, 0>(
              param.out[0], param.outNorm[0], param.in, param.inNorm, idx, face_idx, param.dc.ghostFaceCB[0], param);
        } else {
          const int idx = indexFromFaceIndexExtended<0,nFace,1>(face_idx,param);
          unpackFaceWilsonCore<0, dagger, 1>(
              param.out[1], param.outNorm[1], param.in, param.inNorm, idx, face_idx, param.dc.ghostFaceCB[0], param);
        }
      } else if (dim == 1) {
        const int face_num
            = (param.face_num == 2) ? ((face_idx >= nFace * param.dc.ghostFaceCB[1]) ? 1 : 0) : param.face_num;
        if (param.face_num == 2) face_idx -= face_num * nFace * param.dc.ghostFaceCB[1];

        if (face_num == 0) {
          const int idx = indexFromFaceIndexExtended<1,nFace,0>(face_idx,param);
          unpackFaceWilsonCore<1, dagger, 0>(
              param.out[2], param.outNorm[2], param.in, param.inNorm, idx, face_idx, param.dc.ghostFaceCB[1], param);
        } else {
          const int idx = indexFromFaceIndexExtended<1,nFace,1>(face_idx,param);
          unpackFaceWilsonCore<1, dagger, 1>(
              param.out[3], param.outNorm[3], param.in, param.inNorm, idx, face_idx, param.dc.ghostFaceCB[1], param);
        }
      } else if (dim == 2) {
        const int face_num
            = (param.face_num == 2) ? ((face_idx >= nFace * param.dc.ghostFaceCB[2]) ? 1 : 0) : param.face_num;
        if (param.face_num == 2) face_idx -= face_num * nFace * param.dc.ghostFaceCB[2];

        if (face_num == 0) {
          const int idx = indexFromFaceIndexExtended<2,nFace,0>(face_idx,param);
          unpackFaceWilsonCore<2, dagger, 0>(
              param.out[4], param.outNorm[4], param.in, param.inNorm, idx, face_idx, param.dc.ghostFaceCB[2], param);
        } else {
          const int idx = indexFromFaceIndexExtended<2,nFace,1>(face_idx,param);
          unpackFaceWilsonCore<2, dagger, 1>(
              param.out[5], param.outNorm[5], param.in, param.inNorm, idx, face_idx, param.dc.ghostFaceCB[2], param);
        }
      } else {
        const int face_num
            = (param.face_num == 2) ? ((face_idx >= nFace * param.dc.ghostFaceCB[3]) ? 1 : 0) : param.face_num;
        if (param.face_num == 2) face_idx -= face_num * nFace * param.dc.ghostFaceCB[3];

        if (face_num == 0) {
          const int idx = indexFromFaceIndexExtended<3,nFace,0>(face_idx,param);
          unpackFaceWilsonCore<3, dagger, 0>(
              param.out[6], param.outNorm[6], param.in, param.inNorm, idx, face_idx, param.dc.ghostFaceCB[3], param);
        } else {
          const int idx = indexFromFaceIndexExtended<3,nFace,1>(face_idx,param);
          unpackFaceWilsonCore<3, dagger, 1>(
              param.out[7], param.outNorm[7], param.in, param.inNorm, idx, face_idx, param.dc.ghostFaceCB[3], param);
        }
      }

#ifdef STRIPED
      local_tid += blockDim.x;
      tid += blockDim.x;
#else
      tid += blockDim.x*gridDim.x;
#endif
    } // while tid

  }

#endif // GPU_WILSON_DIRAC || GPU_DOMAIN_WALL_DIRAC


#if defined(GPU_WILSON_DIRAC) || defined(GPU_TWISTED_MASS_DIRAC)


#endif // GPU_WILSON_DIRAC || GPU_DOMAIN_WALL_DIRAC


#if defined(GPU_WILSON_DIRAC) || defined(GPU_TWISTED_MASS_DIRAC)

  // double precision

#endif // GPU_WILSON_DIRAC || GPU_DOMAIN_WALL_DIRAC


#if defined(GPU_WILSON_DIRAC) || defined(GPU_TWISTED_MASS_DIRAC)

  // double precision
#if (defined DIRECT_ACCESS_WILSON_PACK_SPINOR) || (defined FERMI_NO_DBLE_TEX)
#define READ_SPINOR READ_SPINOR_DOUBLE
#define READ_SPINOR_UP READ_SPINOR_DOUBLE_UP
#define READ_SPINOR_DOWN READ_SPINOR_DOUBLE_DOWN
#define SPINORTEX in
#else
#define READ_SPINOR READ_SPINOR_DOUBLE_TEX
#define READ_SPINOR_UP READ_SPINOR_DOUBLE_UP_TEX
#define READ_SPINOR_DOWN READ_SPINOR_DOUBLE_DOWN_TEX
#ifdef USE_TEXTURE_OBJECTS
#define SPINORTEX param.inTex
#else
#define SPINORTEX spinorTexDouble
#endif
#endif
#define WRITE_HALF_SPINOR WRITE_HALF_SPINOR_DOUBLE2
#define SPINOR_DOUBLE
  template <int dim, int dagger, int face_num>
    static inline __device__ void packTwistedFaceWilsonCore(double2 *out, float *outNorm, const double2 *in,
							    const float *inNorm, double a, double b, const int &idx,
							    const int &face_idx, const int &face_volume,
							    PackParam<double2> &param)
  {
    if (dagger) {
#include "wilson_pack_twisted_face_dagger_core.h"
    } else {
#include "wilson_pack_twisted_face_core.h"
    }
  }
#undef READ_SPINOR
#undef READ_SPINOR_UP
#undef READ_SPINOR_DOWN
#undef SPINORTEX
#undef WRITE_HALF_SPINOR
#undef SPINOR_DOUBLE


  // single precision
#ifdef DIRECT_ACCESS_WILSON_PACK_SPINOR
#define READ_SPINOR READ_SPINOR_SINGLE
#define READ_SPINOR_UP READ_SPINOR_SINGLE_UP
#define READ_SPINOR_DOWN READ_SPINOR_SINGLE_DOWN
#define SPINORTEX in
#else
#define READ_SPINOR READ_SPINOR_SINGLE_TEX
#define READ_SPINOR_UP READ_SPINOR_SINGLE_UP_TEX
#define READ_SPINOR_DOWN READ_SPINOR_SINGLE_DOWN_TEX
#ifdef USE_TEXTURE_OBJECTS
#define SPINORTEX param.inTex
#else
#define SPINORTEX spinorTexSingle
#endif
#endif
#define WRITE_HALF_SPINOR WRITE_HALF_SPINOR_FLOAT4
  template <int dim, int dagger, int face_num>
    static inline __device__ void packTwistedFaceWilsonCore(float4 *out, float *outNorm, const float4 *in, const float *inNorm, float a, float b,
							    const int &idx, const int &face_idx,
							    const int &face_volume,
							    const PackParam<float4> &param)
  {
    if (dagger) {
#include "wilson_pack_twisted_face_dagger_core.h"
    } else {
#include "wilson_pack_twisted_face_core.h"
    }
  }
#undef READ_SPINOR
#undef READ_SPINOR_UP
#undef READ_SPINOR_DOWN
#undef SPINORTEX
#undef WRITE_HALF_SPINOR


  // half precision
#ifdef DIRECT_ACCESS_WILSON_PACK_SPINOR
#define READ_SPINOR READ_SPINOR_HALF
#define READ_SPINOR_UP READ_SPINOR_HALF_UP
#define READ_SPINOR_DOWN READ_SPINOR_HALF_DOWN
#define SPINORTEX in
#else
#define READ_SPINOR READ_SPINOR_HALF_TEX
#define READ_SPINOR_UP READ_SPINOR_HALF_UP_TEX
#define READ_SPINOR_DOWN READ_SPINOR_HALF_DOWN_TEX
#ifdef USE_TEXTURE_OBJECTS
#define SPINORTEX param.inTex
#else
#define SPINORTEX spinorTexHalf
#endif
#endif
#define WRITE_HALF_SPINOR WRITE_HALF_SPINOR_SHORT4
  template <int dim, int dagger, int face_num>
    static inline __device__ void packTwistedFaceWilsonCore(short4 *out, float *outNorm, const short4 *in, const float *inNorm, float a, float b,
							    const int &idx, const int &face_idx,
							    const int &face_volume,
							    const PackParam<short4> &param)
  {
    if (dagger) {
#include "wilson_pack_twisted_face_dagger_core.h"
    } else {
#include "wilson_pack_twisted_face_core.h"
    }
  }
#undef READ_SPINOR
#undef READ_SPINOR_UP
#undef READ_SPINOR_DOWN
#undef SPINORTEX
#undef WRITE_HALF_SPINOR

  // quarter precision
#ifdef DIRECT_ACCESS_WILSON_PACK_SPINOR
#define READ_SPINOR READ_SPINOR_QUARTER
#define READ_SPINOR_UP READ_SPINOR_QUARTER_UP
#define READ_SPINOR_DOWN READ_SPINOR_QUARTER_DOWN
#define SPINORTEX in
#else
#define READ_SPINOR READ_SPINOR_QUARTER_TEX
#define READ_SPINOR_UP READ_SPINOR_QUARTER_UP_TEX
#define READ_SPINOR_DOWN READ_SPINOR_QUARTER_DOWN_TEX
#ifdef USE_TEXTURE_OBJECTS
#define SPINORTEX param.inTex
#else
#define SPINORTEX spinorTexHalf
#endif
#endif
#define WRITE_HALF_SPINOR WRITE_HALF_SPINOR_CHAR4
  template <int dim, int dagger, int face_num>
    static inline __device__ void packTwistedFaceWilsonCore(char4 *out, float *outNorm, const char4 *in, const float *inNorm, float a, float b,
                  const int &idx, const int &face_idx, 
                  const int &face_volume, 
                  const PackParam<char4> &param)
  {
    if (dagger) {
#include "wilson_pack_twisted_face_dagger_core.h"
    } else {
#include "wilson_pack_twisted_face_core.h"
    }
  }
#undef READ_SPINOR
#undef READ_SPINOR_UP
#undef READ_SPINOR_DOWN
#undef SPINORTEX
#undef WRITE_HALF_SPINOR

  template <int dagger, typename FloatN, typename Float>
    __global__ void packTwistedFaceWilsonKernel(Float a, Float b, PackParam<FloatN> param)
  {
    const int nFace = 1; // 1 face for Wilson

#ifdef STRIPED
    const int sites_per_block = param.sites_per_block;
    int local_tid = threadIdx.x;
    int tid = sites_per_block * blockIdx.x + local_tid;
#else
    int tid = block_idx(param.swizzle) * blockDim.x + threadIdx.x;
    constexpr int sites_per_block = 1;
    constexpr int local_tid = 0;
#endif

    while ( local_tid < sites_per_block && tid < param.threads ) {

      // determine which dimension we are packing
      int face_idx;
      const int dim = dimFromFaceIndex(face_idx, tid, param);

      // compute where the output is located
      // compute an index into the local volume from the index into the face
      // read spinor, spin-project, and write half spinor to face
      if (dim == 0) {
        // face_num determines which end of the lattice we are packing: 0 = start, 1 = end
        const int face_num = (face_idx >= nFace * param.dc.ghostFaceCB[0]) ? 1 : 0;
        face_idx -= face_num * nFace * param.dc.ghostFaceCB[0];
        if (face_num == 0) {
          const int idx = indexFromFaceIndex<4,QUDA_4D_PC,0,nFace,0>(face_idx,param);
          packTwistedFaceWilsonCore<0, dagger, 0>(param.out[0], param.outNorm[0], param.in, param.inNorm, a, b, idx,
              face_idx, param.dc.ghostFaceCB[0], param);
        } else {
          const int idx = indexFromFaceIndex<4,QUDA_4D_PC,0,nFace,1>(face_idx,param);
          packTwistedFaceWilsonCore<0, dagger, 1>(param.out[1], param.outNorm[1], param.in, param.inNorm, a, b, idx,
              face_idx, param.dc.ghostFaceCB[0], param);
        }
      } else if (dim == 1) {
        const int face_num = (face_idx >= nFace * param.dc.ghostFaceCB[1]) ? 1 : 0;
        face_idx -= face_num * nFace * param.dc.ghostFaceCB[1];
        if (face_num == 0) {
          const int idx = indexFromFaceIndex<4,QUDA_4D_PC,1,nFace,0>(face_idx,param);
          packTwistedFaceWilsonCore<1, dagger, 0>(param.out[2], param.outNorm[2], param.in, param.inNorm, a, b, idx,
              face_idx, param.dc.ghostFaceCB[1], param);
        } else {
          const int idx = indexFromFaceIndex<4,QUDA_4D_PC,1,nFace,1>(face_idx,param);
          packTwistedFaceWilsonCore<1, dagger, 1>(param.out[3], param.outNorm[3], param.in, param.inNorm, a, b, idx,
              face_idx, param.dc.ghostFaceCB[1], param);
        }
      } else if (dim == 2) {
        const int face_num = (face_idx >= nFace * param.dc.ghostFaceCB[2]) ? 1 : 0;
        face_idx -= face_num * nFace * param.dc.ghostFaceCB[2];
        if (face_num == 0) {
          const int idx = indexFromFaceIndex<4,QUDA_4D_PC,2,nFace,0>(face_idx,param);
          packTwistedFaceWilsonCore<2, dagger, 0>(param.out[4], param.outNorm[4], param.in, param.inNorm, a, b, idx,
              face_idx, param.dc.ghostFaceCB[2], param);
        } else {
          const int idx = indexFromFaceIndex<4,QUDA_4D_PC,2,nFace,1>(face_idx,param);
          packTwistedFaceWilsonCore<2, dagger, 1>(param.out[5], param.outNorm[5], param.in, param.inNorm, a, b, idx,
              face_idx, param.dc.ghostFaceCB[2], param);
        }
      } else {
        const int face_num = (face_idx >= nFace * param.dc.ghostFaceCB[3]) ? 1 : 0;
        face_idx -= face_num * nFace * param.dc.ghostFaceCB[3];
        if (face_num == 0) {
          const int idx = indexFromFaceIndex<4,QUDA_4D_PC,3,nFace,0>(face_idx,param);
          packTwistedFaceWilsonCore<3, dagger, 0>(param.out[6], param.outNorm[6], param.in, param.inNorm, a, b, idx,
              face_idx, param.dc.ghostFaceCB[3], param);
        } else {
          const int idx = indexFromFaceIndex<4,QUDA_4D_PC,3,nFace,1>(face_idx,param);
          packTwistedFaceWilsonCore<3, dagger, 1>(param.out[7], param.outNorm[7], param.in, param.inNorm, a, b, idx,
              face_idx, param.dc.ghostFaceCB[3], param);
        }
      }

#ifdef STRIPED
      local_tid += blockDim.x;
      tid += blockDim.x;
#else
      tid += blockDim.x*gridDim.x;
#endif
    } // while tid

  }

#endif // GPU_TWISTED_MASS_DIRAC

  template <typename FloatN, typename Float>
    class PackFace : public Tunable {

  protected:
    void *faces[2*QUDA_MAX_DIM];
    const cudaColorSpinorField *in;
    const int dagger;
    const int parity;
    const int nFace;
    const int dim;
    const int face_num;
    const MemoryLocation location;

    // compute how many threads we need in total for the face packing
    unsigned int threads() const {
      unsigned int threads = 0;
      if(dim < 0){ // if dim is negative, pack all dimensions
        for (int i=0; i<4; i++) {
          if (!commDim[i]) continue;
          if ( i==3 && !getKernelPackT() ) continue;
          threads += 2*nFace*in->GhostFace()[i]; // 2 for forwards and backwards faces
        }
      }else{ // pack only in dim dimension
        if( commDim[dim] && (dim!=3 || getKernelPackT() )){
          threads = nFace*in->GhostFace()[dim];
          if(face_num==2) threads *= 2; // sending data forwards and backwards
        }
      }
      return threads;
    }

    virtual int inputPerSite() const = 0;
    virtual int outputPerSite() const = 0;

    void prepareParam(PackParam<FloatN> &param, TuneParam &tp, int dim=-1, int face_num=2) {
      param.in = (FloatN*)in->V();
      param.inNorm = (float*)in->Norm();
      param.dim = dim;
      param.face_num = face_num;
      param.parity = parity;

#ifdef USE_TEXTURE_OBJECTS
      param.inTex = in->Tex();
      param.inTexNorm = in->TexNorm();
#endif

      param.threads = threads();
      param.sp_stride = in->Stride();

      int prev = -1; // previous dimension that was partitioned
      for (int i=0; i<4; i++) {
        param.threadDimMapLower[i] = 0;
        param.threadDimMapUpper[i] = 0;
        if (!commDim[i]) continue;
        param.threadDimMapLower[i] = (prev>=0 ? param.threadDimMapUpper[prev] : 0);
        param.threadDimMapUpper[i] = param.threadDimMapLower[i] + 2*nFace*in->GhostFace()[i];

	param.out[2*i+0] = static_cast<FloatN*>(faces[2*i+0]);
	param.out[2*i+1] = static_cast<FloatN*>(faces[2*i+1]);

	param.outNorm[2*i+0] = reinterpret_cast<float*>(static_cast<char*>(faces[2*i+0]) + nFace*outputPerSite()*in->GhostFace()[i]*in->Precision());
	param.outNorm[2*i+1] = reinterpret_cast<float*>(static_cast<char*>(faces[2*i+1]) + nFace*outputPerSite()*in->GhostFace()[i]*in->Precision());

        prev=i;
      }

      param.dc = in->getDslashConstant(); // get pre-computed constants

      param.swizzle = tp.aux.x;
      param.sites_per_block = (param.threads + tp.grid.x - 1) / tp.grid.x;
    }

    unsigned int sharedBytesPerThread() const { return 0; }
    unsigned int sharedBytesPerBlock(const TuneParam &param) const { return 0; }

#ifdef STRIPED
    bool tuneGridDim() const { return true; } // If striping, always tune grid dimension
    unsigned int maxGridSize() const {
      if (location & Host) {
	// if zero-copy policy then set a maximum number of blocks to be
	// the 3 * number of dimensions we are communicating
        int nDimComms = 0;
        for (int d=0; d<in->Ndim(); d++) nDimComms += commDim[d];
        return 3*nDimComms;
      } else {
        return Tunable::maxGridSize();
      }
    } // use no more than a quarter of the GPU
    unsigned int minGridSize() const {
      if (location & Host) {
	// if zero-copy policy then set a maximum number of blocks to be
	// the 1 * number of dimensions we are communicating
        int nDimComms = 0;
        for (int d=0; d<in->Ndim(); d++) nDimComms += commDim[d];
        return nDimComms;
      } else {
        return Tunable::minGridSize();
      }
    }
#else
    bool tuneGridDim() const { return location & Host; } // only tune grid dimension if doing zero-copy writing
    unsigned int maxGridSize() const
    {
      return tuneGridDim() ? deviceProp.multiProcessorCount / 4 : Tunable::maxGridSize();
    } // use no more than a quarter of the GPU
#endif

    bool tuneAuxDim() const { return true; } // Do tune the aux dimensions.
    unsigned int minThreads() const { return threads(); }

    void fillAux() {
      strcpy(aux,"policy_kernel,");
      strcat(aux, in->AuxString());
      char comm[5];
      comm[0] = (commDim[0] ? '1' : '0');
      comm[1] = (commDim[1] ? '1' : '0');
      comm[2] = (commDim[2] ? '1' : '0');
      comm[3] = (commDim[3] ? '1' : '0');
      comm[4] = '\0'; strcat(aux,",comm=");
      strcat(aux,comm);
      strcat(aux,comm_dim_topology_string());
      if (getKernelPackT()) { strcat(aux,",kernelPackT"); }
      switch (nFace) {
      case 1: strcat(aux,",nFace=1,"); break;
      case 3: strcat(aux,",nFace=3,"); break;
      default: errorQuda("Number of faces not supported");
      }

      // label the locations we are packing to
      // location lable is nonp2p-p2p
      switch ((int)location) {
      case Device|Remote: strcat(aux,"device-remote"); break;
      case   Host|Remote: strcat(aux,  "host-remote"); break;
      case        Device: strcat(aux,"device-device"); break;
      case          Host: strcat(aux, comm_peer2peer_enabled_global() ? "host-device" : "host-host"); break;
      default: errorQuda("Unknown pack target location %d\n", location);
      }

    }

  public:
    PackFace(void *faces_[], const cudaColorSpinorField *in, MemoryLocation location,
	     const int dagger, const int parity, const int nFace, const int dim=-1, const int face_num=2)
      : in(in), dagger(dagger),
	parity(parity), nFace(nFace), dim(dim), face_num(face_num), location(location)
    {
      memcpy(faces, faces_, 2*QUDA_MAX_DIM*sizeof(void*));
      fillAux();
#ifndef USE_TEXTURE_OBJECTS
      bindSpinorTex<FloatN>(in);
#endif
    }

    virtual ~PackFace() {
#ifndef USE_TEXTURE_OBJECTS
      unbindSpinorTex<FloatN>(in);
#endif
    }

    bool tuneSharedBytes() const { return location & Host ? false : Tunable::tuneSharedBytes(); }

    bool advanceAux(TuneParam &param) const
    {
#ifdef SWIZZLE
      if ( location & Remote ) {  // only swizzling if we're doing remote writing
        if (param.aux.x < (int)maxGridSize()) {
          param.aux.x++;
          return true;
        } else {
          param.aux.x = 1;
          return false;
        }
      } else {
        return false;
      }
#else
      return false;
#endif
    }

    void initTuneParam(TuneParam &param) const {
      Tunable::initTuneParam(param);
      param.aux.x = 1; // swizzle factor
      // if doing a zero-copy policy then ensure that each thread block
      // runs exclusively on a given SM - this is to ensure quality of
      // service for the packing kernel when running concurrently.
      if (location & Host) param.shared_bytes = deviceProp.sharedMemPerBlock / 2 + 1;
    }

    void defaultTuneParam(TuneParam &param) const {
      Tunable::defaultTuneParam(param);
      param.aux.x = 1; // swizzle factor
    }

    long long flops() const { return outputPerSite()*this->threads(); }

    virtual int tuningIter() const { return 3; }

    virtual TuneKey tuneKey() const { return TuneKey(in->VolString(), typeid(*this).name(), aux); }

    virtual void apply(const cudaStream_t &stream) = 0;

    long long bytes() const {
      size_t faceBytes = (inputPerSite() + outputPerSite())*this->threads()*sizeof(((FloatN*)0)->x);
      if (sizeof(((FloatN*)0)->x) == QUDA_HALF_PRECISION)
        faceBytes += 2*this->threads()*sizeof(float); // 2 is from input and output
      return faceBytes;
    }
  };

  template <typename FloatN, typename Float>
    class PackFaceWilson : public PackFace<FloatN, Float> {

  private:

    int inputPerSite() const { return 24; } // input is full spinor
    int outputPerSite() const { return 12; } // output is spin projected

  public:
    PackFaceWilson(void *faces[], const cudaColorSpinorField *in, MemoryLocation location,
		   const int dagger, const int parity)
      : PackFace<FloatN, Float>(faces, in, location, dagger, parity, 1) { }
    virtual ~PackFaceWilson() { }

    void apply(const cudaStream_t &stream) {
      TuneParam tp = tuneLaunch(*this, getTuning(), getVerbosity());

#ifdef GPU_WILSON_DIRAC
      static PackParam<FloatN> param;
      this->prepareParam(param,tp);

      void *args[] = { &param };
      void (*func)(PackParam<FloatN>) = this->dagger ? &(packFaceWilsonKernel<1,FloatN>) : &(packFaceWilsonKernel<0,FloatN>);
      qudaLaunchKernel( (const void*)func, tp.grid, tp.block, args, tp.shared_bytes, stream);
#else
      errorQuda("Wilson face packing kernel is not built");
#endif
    }

  };

  void packFaceWilson(void *ghost_buf[], cudaColorSpinorField &in, MemoryLocation location,
		      const int dagger, const int parity, const cudaStream_t &stream) {

    switch(in.Precision()) {
    case QUDA_DOUBLE_PRECISION:
      {
        PackFaceWilson<double2, double> pack(ghost_buf, &in, location, dagger, parity);
        pack.apply(stream);
      }
      break;
    case QUDA_SINGLE_PRECISION:
      {
        PackFaceWilson<float4, float> pack(ghost_buf, &in, location, dagger, parity);
        pack.apply(stream);
      }
      break;
    case QUDA_HALF_PRECISION:
      {
        PackFaceWilson<short4, float> pack(ghost_buf, &in, location, dagger, parity);
        pack.apply(stream);
      }
      break;
    case QUDA_QUARTER_PRECISION:
      {
        PackFaceWilson<char4, float> pack(ghost_buf, &in, location, dagger, parity);
        pack.apply(stream);
      }
      break;
    default:
      errorQuda("Precision %d not supported", in.Precision());
    }
  }

  template <typename FloatN, typename Float>
    class PackFaceTwisted : public PackFace<FloatN, Float> {

  private:

    int inputPerSite() const { return 24; } // input is full spinor
    int outputPerSite() const { return 12; } // output is spin projected
    Float a;
    Float b;

  public:
    PackFaceTwisted(void *faces[], const cudaColorSpinorField *in, MemoryLocation location,
		    const int dagger, const int parity, Float a, Float b)
      : PackFace<FloatN, Float>(faces, in, location, dagger, parity, 1), a(a), b(b) { }
    virtual ~PackFaceTwisted() { }

    void apply(const cudaStream_t &stream) {
      TuneParam tp = tuneLaunch(*this, getTuning(), getVerbosity());

#ifdef GPU_TWISTED_MASS_DIRAC
      static PackParam<FloatN> param;
      this->prepareParam(param,tp);
      void *args[] = { &a, &b, &param };
      void (*func)(Float,Float,PackParam<FloatN>) = this->dagger ? &(packTwistedFaceWilsonKernel<1,FloatN,Float>) : &(packTwistedFaceWilsonKernel<0,FloatN,Float>);
      cudaLaunchKernel( (const void*)func, tp.grid, tp.block, args, tp.shared_bytes, stream);
#else
      errorQuda("Twisted face packing kernel is not built");
#endif
    }

    long long flops() const { return outputPerSite()*this->threads(); }
  };

  //!
  void packTwistedFaceWilson(void *ghost_buf[], cudaColorSpinorField &in, MemoryLocation location, const int dagger,
			     const int parity, const double a, const double b, const cudaStream_t &stream) {

    switch(in.Precision()) {
    case QUDA_DOUBLE_PRECISION:
      {
        PackFaceTwisted<double2, double> pack(ghost_buf, &in, location, dagger, parity, a, b);
        pack.apply(stream);
      }
      break;
    case QUDA_SINGLE_PRECISION:
      {
        PackFaceTwisted<float4, float> pack(ghost_buf, &in, location, dagger, parity, (float)a, (float)b);
        pack.apply(stream);
      }
      break;
    case QUDA_HALF_PRECISION:
      {
        PackFaceTwisted<short4, float> pack(ghost_buf, &in, location, dagger, parity, (float)a, (float)b);
        pack.apply(stream);
      }
      break;
    case QUDA_QUARTER_PRECISION:
      {
        PackFaceTwisted<char4, float> pack(ghost_buf, &in, location, dagger, parity, (float)a, (float)b);
        pack.apply(stream);
      }
      break;
    default:
      errorQuda("Precision %d not supported", in.Precision());
    }
  }

#if (defined GPU_STAGGERED_DIRAC)

#ifdef USE_TEXTURE_OBJECTS
#define SPINORTEXDOUBLE param.inTex
#define SPINORTEXSINGLE param.inTex
#define SPINORTEXHALF param.inTex
#define SPINORTEXHALFNORM param.inTexNorm
#define SPINORTEXQUARTER param.inTex
#define SPINORTEXQUARTERNORM param.inTexNorm
#else
#define SPINORTEXDOUBLE spinorTexDouble
#define SPINORTEXSINGLE spinorTexSingle2
#define SPINORTEXHALF spinorTexHalf2
#define SPINORTEXHALFNORM spinorTexHalf2Norm
#define SPINORTEXQUARTER spinorTexQuarter2
#define SPINORTEXQUARTERNORM spinorTexQuarter2Norm
#endif

  template <typename Float2>
    __device__ void packFaceStaggeredCore(Float2 *out, float *outNorm, const int out_idx,
					  const int out_stride, const Float2 *in, const float *inNorm,
					  const int in_idx, const int in_stride) {
    out[out_idx + 0*out_stride] = in[in_idx + 0*in_stride];
    out[out_idx + 1*out_stride] = in[in_idx + 1*in_stride];
    out[out_idx + 2*out_stride] = in[in_idx + 2*in_stride];
  }
  template<>
    __device__ void packFaceStaggeredCore(short2 *out, float *outNorm, const int out_idx,
					  const int out_stride, const short2 *in, const float *inNorm,
					  const int in_idx, const int in_stride) {
    out[out_idx + 0*out_stride] = in[in_idx + 0*in_stride];
    out[out_idx + 1*out_stride] = in[in_idx + 1*in_stride];
    out[out_idx + 2*out_stride] = in[in_idx + 2*in_stride];
    outNorm[out_idx] = inNorm[in_idx];
  }
  template<> 
    __device__ void packFaceStaggeredCore(char2 *out, float *outNorm, const int out_idx, 
            const int out_stride, const char2 *in, const float *inNorm, 
            const int in_idx, const int in_stride) {
    out[out_idx + 0*out_stride] = in[in_idx + 0*in_stride];
    out[out_idx + 1*out_stride] = in[in_idx + 1*in_stride];
    out[out_idx + 2*out_stride] = in[in_idx + 2*in_stride];
    outNorm[out_idx] = inNorm[in_idx];
  }

#if (defined DIRECT_ACCESS_PACK) || (defined FERMI_NO_DBLE_TEX)
  template <typename Float2>
    __device__ void packFaceStaggeredCore(Float2 *out, float *outNorm, const int out_idx,
					  const int out_stride, const Float2 *in, const float *inNorm,
					  const int in_idx, const PackParam<double2> &param) {
    out[out_idx + 0*out_stride] = in[in_idx + 0*param.sp_stride];
    out[out_idx + 1*out_stride] = in[in_idx + 1*param.sp_stride];
    out[out_idx + 2*out_stride] = in[in_idx + 2*param.sp_stride];
  }
  template<>
    __device__ void packFaceStaggeredCore(short2 *out, float *outNorm, const int out_idx,
					  const int out_stride, const short2 *in, const float *inNorm,
					  const int in_idx, const PackParam<double2> &param) {
    out[out_idx + 0*out_stride] = in[in_idx + 0*param.sp_stride];
    out[out_idx + 1*out_stride] = in[in_idx + 1*param.sp_stride];
    out[out_idx + 2*out_stride] = in[in_idx + 2*param.sp_stride];
    outNorm[out_idx] = inNorm[in_idx];
  }
  template<> 
    __device__ void packFaceStaggeredCore(char2 *out, float *outNorm, const int out_idx, 
            const int out_stride, const char2 *in, const float *inNorm, 
            const int in_idx, const PackParam<double2> &param) {
    out[out_idx + 0*out_stride] = in[in_idx + 0*param.sp_stride];
    out[out_idx + 1*out_stride] = in[in_idx + 1*param.sp_stride];
    out[out_idx + 2*out_stride] = in[in_idx + 2*param.sp_stride];
    outNorm[out_idx] = inNorm[in_idx];
  }


#else
  __device__ void packFaceStaggeredCore(double2 *out, float *outNorm, const int out_idx,
					const int out_stride, const double2 *in, const float *inNorm,
					const int in_idx, const PackParam<double2> &param) {
    out[out_idx + 0*out_stride] = fetch_double2(SPINORTEXDOUBLE, in_idx + 0*param.sp_stride);
    out[out_idx + 1*out_stride] = fetch_double2(SPINORTEXDOUBLE, in_idx + 1*param.sp_stride);
    out[out_idx + 2*out_stride] = fetch_double2(SPINORTEXDOUBLE, in_idx + 2*param.sp_stride);
  }

  __device__ void packFaceStaggeredCore(float2 *out, float *outNorm, const int out_idx,
					const int out_stride, const float2 *in,
					const float *inNorm, const int in_idx,
					const PackParam<float2> &param) {
    out[out_idx + 0 * out_stride] = tex1Dfetch<float2>(SPINORTEXSINGLE, in_idx + 0 * param.sp_stride);
    out[out_idx + 1 * out_stride] = tex1Dfetch<float2>(SPINORTEXSINGLE, in_idx + 1 * param.sp_stride);
    out[out_idx + 2 * out_stride] = tex1Dfetch<float2>(SPINORTEXSINGLE, in_idx + 2 * param.sp_stride);
  }

  // this is rather dumb: undoing the texture load because cudaNormalizedReadMode is used
  // should really bind to an appropriate texture instead of reusing
  inline __device__ short2 float22short2(float c, float2 a) {
    return make_short2((short)(a.x*(c*fixedMaxValue<short>::value)), (short)(a.y*(c*fixedMaxValue<short>::value)));
  }
  
  inline __device__ char2 float22char2(float c, float2 a) {
    return make_char2((char)(a.x*(c*fixedMaxValue<char>::value)), (char)(a.y*(c*fixedMaxValue<char>::value)));
  }

  __device__ void packFaceStaggeredCore(short2 *out, float *outNorm, const int out_idx,
					const int out_stride, const short2 *in,
					const float *inNorm, const int in_idx,
					const PackParam<short2> &param) {
    out[out_idx + 0 * out_stride] = float22short2(1.0f, tex1Dfetch<float2>(SPINORTEXHALF, in_idx + 0 * param.sp_stride));
    out[out_idx + 1 * out_stride] = float22short2(1.0f, tex1Dfetch<float2>(SPINORTEXHALF, in_idx + 1 * param.sp_stride));
    out[out_idx + 2 * out_stride] = float22short2(1.0f, tex1Dfetch<float2>(SPINORTEXHALF, in_idx + 2 * param.sp_stride));
    outNorm[out_idx] = tex1Dfetch<float>(SPINORTEXHALFNORM, in_idx);
  }

  __device__ void packFaceStaggeredCore(char2 *out, float *outNorm, const int out_idx, 
          const int out_stride, const char2 *in, 
          const float *inNorm, const int in_idx, 
          const PackParam<char2> &param) {
    out[out_idx + 0 * out_stride]
        = float22char2(1.0f, tex1Dfetch<float2>(SPINORTEXQUARTER, in_idx + 0 * param.sp_stride));
    out[out_idx + 1 * out_stride]
        = float22char2(1.0f, tex1Dfetch<float2>(SPINORTEXQUARTER, in_idx + 1 * param.sp_stride));
    out[out_idx + 2 * out_stride]
        = float22char2(1.0f, tex1Dfetch<float2>(SPINORTEXQUARTER, in_idx + 2 * param.sp_stride));
    outNorm[out_idx] = tex1Dfetch<float>(SPINORTEXQUARTERNORM, in_idx);
  }
#endif


  template <typename FloatN, int nFace>
    __global__ void packFaceStaggeredKernel(PackParam<FloatN> param)
  {

#ifdef STRIPED
    const int sites_per_block = param.sites_per_block;
    int local_tid = threadIdx.x;
    int tid = sites_per_block * blockIdx.x + local_tid;
#else
    int tid = block_idx(param.swizzle) * blockDim.x + threadIdx.x;
    constexpr int sites_per_block = 1;
    constexpr int local_tid = 0;
#endif

    while ( local_tid < sites_per_block && tid < param.threads ) {

      const int Ls = param.dc.X[4];

      // determine which dimension we are packing
      int face_idx;
      const int dim = dimFromFaceIndex(face_idx, tid, param);

      // compute where the output is located
      // compute an index into the local volume from the index into the face
      // read spinor and write to face
      if (dim == 0) {
        // face_num determines which end of the lattice we are packing: 0 = start, 1 = end
        const int face_num
            = (param.face_num == 2) ? ((face_idx >= Ls * nFace * param.dc.ghostFaceCB[0]) ? 1 : 0) : param.face_num;
        if (param.face_num == 2) face_idx -= face_num * Ls * nFace * param.dc.ghostFaceCB[0];
        if (face_num == 0) {
          const int idx = indexFromFaceIndexStaggered<0,nFace,0>(face_idx,param);
          packFaceStaggeredCore(param.out[0], param.outNorm[0], face_idx, Ls * nFace * param.dc.ghostFaceCB[0],
              param.in, param.inNorm, idx, param);
        } else {
          const int idx = indexFromFaceIndexStaggered<0,nFace,1>(face_idx,param);
          packFaceStaggeredCore(param.out[1], param.outNorm[1], face_idx, Ls * nFace * param.dc.ghostFaceCB[0],
              param.in, param.inNorm, idx, param);
        }
      } else if (dim == 1) {
        const int face_num
            = (param.face_num == 2) ? ((face_idx >= Ls * nFace * param.dc.ghostFaceCB[1]) ? 1 : 0) : param.face_num;
        if (param.face_num == 2) face_idx -= face_num * Ls * nFace * param.dc.ghostFaceCB[1];
        if (face_num == 0) {
          const int idx = indexFromFaceIndexStaggered<1,nFace,0>(face_idx,param);
          packFaceStaggeredCore(param.out[2], param.outNorm[2], face_idx, Ls * nFace * param.dc.ghostFaceCB[1],
              param.in, param.inNorm, idx, param);
        } else {
          const int idx = indexFromFaceIndexStaggered<1,nFace,1>(face_idx,param);
          packFaceStaggeredCore(param.out[3], param.outNorm[3], face_idx, Ls * nFace * param.dc.ghostFaceCB[1],
              param.in, param.inNorm, idx, param);
        }
      } else if (dim == 2) {
        const int face_num
            = (param.face_num == 2) ? ((face_idx >= Ls * nFace * param.dc.ghostFaceCB[2]) ? 1 : 0) : param.face_num;
        if (param.face_num == 2) face_idx -= face_num * Ls * nFace * param.dc.ghostFaceCB[2];
        if (face_num == 0) {
          const int idx = indexFromFaceIndexStaggered<2,nFace,0>(face_idx,param);
          packFaceStaggeredCore(param.out[4], param.outNorm[4], face_idx, Ls * nFace * param.dc.ghostFaceCB[2],
              param.in, param.inNorm, idx, param);
        } else {
          const int idx = indexFromFaceIndexStaggered<2,nFace,1>(face_idx,param);
          packFaceStaggeredCore(param.out[5], param.outNorm[5], face_idx, Ls * nFace * param.dc.ghostFaceCB[2],
              param.in, param.inNorm, idx, param);
        }
      } else {
        const int face_num
            = (param.face_num == 2) ? ((face_idx >= Ls * nFace * param.dc.ghostFaceCB[3]) ? 1 : 0) : param.face_num;
        if (param.face_num == 2) face_idx -= face_num * Ls * nFace * param.dc.ghostFaceCB[3];
        if (face_num == 0) {
          const int idx = indexFromFaceIndexStaggered<3,nFace,0>(face_idx,param);
          packFaceStaggeredCore(param.out[6], param.outNorm[6], face_idx, Ls * nFace * param.dc.ghostFaceCB[3],
              param.in, param.inNorm, idx, param);
        } else {
          const int idx = indexFromFaceIndexStaggered<3,nFace,1>(face_idx,param);
          packFaceStaggeredCore(param.out[7], param.outNorm[7], face_idx, Ls * nFace * param.dc.ghostFaceCB[3],
              param.in, param.inNorm, idx, param);
        }
      }

#ifdef STRIPED
      local_tid += blockDim.x;
      tid += blockDim.x;
#else
      tid += blockDim.x*gridDim.x;
#endif
    } // while tid

  }


  template <typename FloatN, int nFace>
    __global__ void packFaceExtendedStaggeredKernel(PackExtendedParam<FloatN> param)
  {

#ifdef STRIPED
    const int sites_per_block = param.sites_per_block;
    int local_tid = threadIdx.x;
    int tid = sites_per_block * blockIdx.x + local_tid;
#else
    int tid = block_idx(param.swizzle) * blockDim.x + threadIdx.x;
    constexpr int sites_per_block = 1;
    constexpr int local_tid = 0;
#endif

    while ( local_tid < sites_per_block && tid < param.threads ) {

      // determine which dimension we are packing
      int face_idx;
      const int dim = dimFromFaceIndex(face_idx, tid, param);

      // compute where the output is located
      // compute an index into the local volume from the index into the face
      // read spinor and write half spinor to face
      if (dim == 0) {
        // face_num determines which end of the lattice we are packing: 0 = start, 1 = end
        // if param.face_num==2 pack both the start and the end, otherwise pack the region of the
        // lattice specified by param.face_num
        const int face_num
            = (param.face_num == 2) ? ((face_idx >= nFace * param.dc.ghostFaceCB[0]) ? 1 : 0) : param.face_num;
        if (param.face_num == 2) face_idx -= face_num * nFace * param.dc.ghostFaceCB[0];
        if (face_num == 0) {
          const int idx = indexFromFaceIndexExtendedStaggered<0,nFace,0>(face_idx,param);
          packFaceStaggeredCore(param.out[0], param.outNorm[0], face_idx, nFace * param.dc.ghostFaceCB[0], param.in,
              param.inNorm, idx, param);
        } else {
          const int idx = indexFromFaceIndexExtendedStaggered<0,nFace,1>(face_idx,param);
          packFaceStaggeredCore(param.out[1], param.outNorm[1], face_idx, nFace * param.dc.ghostFaceCB[0], param.in,
              param.inNorm, idx, param);
        }
      } else if (dim == 1) {
        const int face_num
            = (param.face_num == 2) ? ((face_idx >= nFace * param.dc.ghostFaceCB[1]) ? 1 : 0) : param.face_num;
        if (param.face_num == 2) face_idx -= face_num * nFace * param.dc.ghostFaceCB[1];
        if (face_num == 0) {
          const int idx = indexFromFaceIndexExtendedStaggered<1,nFace,0>(face_idx,param);
          packFaceStaggeredCore(param.out[2], param.outNorm[2], face_idx, nFace * param.dc.ghostFaceCB[1], param.in,
              param.inNorm, idx, param);
        } else {
          const int idx = indexFromFaceIndexExtendedStaggered<1,nFace,1>(face_idx,param);
          packFaceStaggeredCore(param.out[3], param.outNorm[3], face_idx, nFace * param.dc.ghostFaceCB[1], param.in,
              param.inNorm, idx, param);
        }
      } else if (dim == 2) {
        const int face_num
            = (param.face_num == 2) ? ((face_idx >= nFace * param.dc.ghostFaceCB[2]) ? 1 : 0) : param.face_num;
        if (param.face_num == 2) face_idx -= face_num * nFace * param.dc.ghostFaceCB[2];
        if (face_num == 0) {
          const int idx = indexFromFaceIndexExtendedStaggered<2,nFace,0>(face_idx,param);
          packFaceStaggeredCore(param.out[4], param.outNorm[4], face_idx, nFace * param.dc.ghostFaceCB[2], param.in,
              param.inNorm, idx, param);
        } else {
          const int idx = indexFromFaceIndexExtendedStaggered<2,nFace,1>(face_idx,param);
          packFaceStaggeredCore(param.out[5], param.outNorm[5], face_idx, nFace * param.dc.ghostFaceCB[2], param.in,
              param.inNorm, idx, param);
        }
      } else {
        const int face_num
            = (param.face_num == 2) ? ((face_idx >= nFace * param.dc.ghostFaceCB[3]) ? 1 : 0) : param.face_num;
        if (param.face_num == 2) face_idx -= face_num * nFace * param.dc.ghostFaceCB[3];
        if (face_num == 0) {
          const int idx = indexFromFaceIndexExtendedStaggered<3,nFace,0>(face_idx,param);
          packFaceStaggeredCore(param.out[6], param.outNorm[6], face_idx, nFace * param.dc.ghostFaceCB[3], param.in,
              param.inNorm, idx, param);
        } else {
          const int idx = indexFromFaceIndexExtendedStaggered<3,nFace,1>(face_idx,param);
          packFaceStaggeredCore(param.out[7], param.outNorm[7], face_idx, nFace * param.dc.ghostFaceCB[3], param.in,
              param.inNorm, idx, param);
        }
      }

#ifdef STRIPED
      local_tid += blockDim.x;
      tid += blockDim.x;
#else
      tid += blockDim.x*gridDim.x;
#endif
    } // while tid

  }


  template <typename FloatN, int nFace>
    __global__ void unpackFaceExtendedStaggeredKernel(PackExtendedParam<FloatN> param)
  {

#ifdef STRIPED
    const int sites_per_block = param.sites_per_block;
    int local_tid = threadIdx.x;
    int tid = sites_per_block * blockIdx.x + local_tid;
#else
    int tid = block_idx(param.swizzle) * blockDim.x + threadIdx.x;
    constexpr int sites_per_block = 1;
    constexpr int local_tid = 0;
#endif

    while ( local_tid < sites_per_block && tid < param.threads ) {

      // determine which dimension we are packing
      int face_idx;
      const int dim = dimFromFaceIndex(face_idx, tid, param);

      // compute where the output is located
      // compute an index into the local volume from the index into the face
      // read spinor, spin-project, and write half spinor to face
      if (dim == 0) {
        // face_num determines which end of the lattice we are packing: 0 = start, 1 = end
        // if param.face_num==2 pack both the start and the end, otherwist pack the region of the
        // lattice specified by param.face_num
        const int face_num
            = (param.face_num == 2) ? ((face_idx >= nFace * param.dc.ghostFaceCB[0]) ? 1 : 0) : param.face_num;
        if (param.face_num == 2) face_idx -= face_num * nFace * param.dc.ghostFaceCB[0];
        if (face_num == 0) {
          const int idx = indexFromFaceIndexExtendedStaggered<0,nFace,0>(face_idx,param);
          packFaceStaggeredCore(param.in, param.inNorm, idx, param.sp_stride, param.out[0], param.outNorm[0], face_idx,
              nFace * param.dc.ghostFaceCB[0]);
        } else {
          const int idx = indexFromFaceIndexExtendedStaggered<0,nFace,1>(face_idx,param);
          packFaceStaggeredCore(param.in, param.inNorm, idx, param.sp_stride, param.out[1], param.outNorm[1], face_idx,
              nFace * param.dc.ghostFaceCB[0]);
        }
      } else if (dim == 1) {
        const int face_num
            = (param.face_num == 2) ? ((face_idx >= nFace * param.dc.ghostFaceCB[1]) ? 1 : 0) : param.face_num;
        if (param.face_num == 2) face_idx -= face_num * nFace * param.dc.ghostFaceCB[1];
        if (face_num == 0) {
          const int idx = indexFromFaceIndexExtendedStaggered<1,nFace,0>(face_idx,param);
          packFaceStaggeredCore(param.in, param.inNorm, idx, param.sp_stride, param.out[2], param.outNorm[2], face_idx,
              nFace * param.dc.ghostFaceCB[1]);
        } else {
          const int idx = indexFromFaceIndexExtendedStaggered<1,nFace,1>(face_idx,param);
          packFaceStaggeredCore(param.in, param.inNorm, idx, param.sp_stride, param.out[3], param.outNorm[3], face_idx,
              nFace * param.dc.ghostFaceCB[1]);
        }
      } else if (dim == 2) {
        const int face_num
            = (param.face_num == 2) ? ((face_idx >= nFace * param.dc.ghostFaceCB[2]) ? 1 : 0) : param.face_num;
        if (param.face_num == 2) face_idx -= face_num * nFace * param.dc.ghostFaceCB[2];
        if (face_num == 0) {
          const int idx = indexFromFaceIndexExtendedStaggered<2,nFace,0>(face_idx,param);
          packFaceStaggeredCore(param.in, param.inNorm, idx, param.sp_stride, param.out[4], param.outNorm[4], face_idx,
              nFace * param.dc.ghostFaceCB[2]);
        } else {
          const int idx = indexFromFaceIndexExtendedStaggered<2,nFace,1>(face_idx,param);
          packFaceStaggeredCore(param.in, param.inNorm, idx, param.sp_stride, param.out[5], param.outNorm[5], face_idx,
              nFace * param.dc.ghostFaceCB[2]);
        }
      } else {
        const int face_num
            = (param.face_num == 2) ? ((face_idx >= nFace * param.dc.ghostFaceCB[3]) ? 1 : 0) : param.face_num;
        if (param.face_num == 2) face_idx -= face_num * nFace * param.dc.ghostFaceCB[3];
        if (face_num == 0) {
          const int idx = indexFromFaceIndexExtendedStaggered<3,nFace,0>(face_idx,param);
          packFaceStaggeredCore(param.in, param.inNorm, idx, param.sp_stride, param.out[6], param.outNorm[6], face_idx,
              nFace * param.dc.ghostFaceCB[3]);
        } else {
          const int idx = indexFromFaceIndexExtendedStaggered<3,nFace,1>(face_idx,param);
          packFaceStaggeredCore(param.in, param.inNorm, idx, param.sp_stride, param.out[7], param.outNorm[7], face_idx,
              nFace * param.dc.ghostFaceCB[3]);
        }
      }

#ifdef STRIPED
      local_tid += blockDim.x;
      tid += blockDim.x;
#else
      tid += blockDim.x*gridDim.x;
#endif
    } // while tid

  }


#undef SPINORTEXDOUBLE
#undef SPINORTEXSINGLE
#undef SPINORTEXHALF
#undef SPINORTEXQUARTER

#endif // GPU_STAGGERED_DIRAC


  template <typename FloatN, typename Float>
    class PackFaceStaggered : public PackFace<FloatN, Float> {

  private:
    const int* R; // boundary dimensions for extended field
    const bool unpack;

    int inputPerSite() const { return 6; } // input is full spinor
    int outputPerSite() const { return 6; } // output is full spinor


  public:
    PackFaceStaggered(void *faces[], const cudaColorSpinorField *in, MemoryLocation location,
		      const int nFace, const int dagger, const int parity,
		      const int dim, const int face_num, const int* R=NULL, const bool unpack=false)
      : PackFace<FloatN, Float>(faces, in, location, dagger, parity, nFace, dim, face_num), R(R), unpack(unpack) { }
    virtual ~PackFaceStaggered() { }

    void apply(const cudaStream_t &stream) {
      TuneParam tp = tuneLaunch(*this, getTuning(), getVerbosity());

#if (defined GPU_STAGGERED_DIRAC)

      static PackParam<FloatN> param;
      this->prepareParam(param,tp,this->dim,this->face_num);
      if(!R){
        void *args[] = { &param };
        void (*func)(PackParam<FloatN>) = PackFace<FloatN,Float>::nFace==1 ? &(packFaceStaggeredKernel<FloatN,1>) : &(packFaceStaggeredKernel<FloatN,3>);
        cudaLaunchKernel( (const void*)func, tp.grid, tp.block, args, tp.shared_bytes, stream);
      }else{ // R!=NULL => this is an extended field
        PackExtendedParam<FloatN> extendedParam(param);
        if(!unpack){
          for(int d=0; d<QUDA_MAX_DIM; ++d) extendedParam.R[d] = R[d];
	  switch(PackFace<FloatN,Float>::nFace){
	  case 1:
	    packFaceExtendedStaggeredKernel<FloatN,1><<<tp.grid, tp.block, tp.shared_bytes, stream>>>(extendedParam);
	    break;

	  case 2:
	    packFaceExtendedStaggeredKernel<FloatN,2><<<tp.grid, tp.block, tp.shared_bytes, stream>>>(extendedParam);
	    break;

	  case 3:
	    packFaceExtendedStaggeredKernel<FloatN,3><<<tp.grid, tp.block, tp.shared_bytes, stream>>>(extendedParam);
	    break;

	  case 4:
	    packFaceExtendedStaggeredKernel<FloatN,4><<<tp.grid, tp.block, tp.shared_bytes, stream>>>(extendedParam);
	    break;

	  default:
	    errorQuda("Unsupported boundary width");
	    break;
	  }
        }else{ // extended field unpack
	  switch(PackFace<FloatN,Float>::nFace){
	  case 1:
	    unpackFaceExtendedStaggeredKernel<FloatN,1><<<tp.grid, tp.block, tp.shared_bytes, stream>>>(extendedParam);
	    break;

	  case 2:
	    unpackFaceExtendedStaggeredKernel<FloatN,2><<<tp.grid, tp.block, tp.shared_bytes, stream>>>(extendedParam);
	    break;

	  case 3:
	    unpackFaceExtendedStaggeredKernel<FloatN,3><<<tp.grid, tp.block, tp.shared_bytes, stream>>>(extendedParam);
	    break;

	  case 4:
	    unpackFaceExtendedStaggeredKernel<FloatN,4><<<tp.grid, tp.block, tp.shared_bytes, stream>>>(extendedParam);
	    break;

	  default:
	    errorQuda("Unsupported boundary width");
	    break;
	  }
	}
      }
#else
      errorQuda("Staggered face packing kernel is not built");
#endif
    }

    long long flops() const { return 0; }
  };


  void packFaceStaggered(void *ghost_buf[], cudaColorSpinorField &in, MemoryLocation location, int nFace,
			 int dagger, int parity, const int dim, const int face_num, const cudaStream_t &stream) {

    switch(in.Precision()) {
    case QUDA_DOUBLE_PRECISION:
      {
        PackFaceStaggered<double2, double> pack(ghost_buf, &in, location, nFace, dagger, parity, dim, face_num);
        pack.apply(stream);
      }
      break;
    case QUDA_SINGLE_PRECISION:
      {
        PackFaceStaggered<float2, float> pack(ghost_buf, &in, location, nFace, dagger, parity, dim, face_num);
        pack.apply(stream);
      }
      break;
    case QUDA_HALF_PRECISION:
      {
        PackFaceStaggered<short2, float> pack(ghost_buf, &in, location, nFace, dagger, parity, dim, face_num);
        pack.apply(stream);
      }
      break;
    case QUDA_QUARTER_PRECISION:
      {
        PackFaceStaggered<char2, float> pack(ghost_buf, &in, location, nFace, dagger, parity, dim, face_num);
        pack.apply(stream);
      }
      break;
    default:
      errorQuda("Precision %d not supported", in.Precision());
    }
  }

  void packFaceExtendedStaggered(void *buffer[], cudaColorSpinorField &field, MemoryLocation location,  const int nFace, const int R[],
				 int dagger, int parity, const int dim, const int face_num, const cudaStream_t &stream, bool unpack=false)
  {
    switch(field.Precision()){
    case QUDA_DOUBLE_PRECISION:
      {
        PackFaceStaggered<double2,double> pack(buffer, &field, location, nFace, dagger, parity, dim, face_num, R, unpack);
        pack.apply(stream);
      }
      break;
    case QUDA_SINGLE_PRECISION:
      {
        PackFaceStaggered<float2,float> pack(buffer, &field, location, nFace, dagger, parity, dim, face_num, R, unpack);
        pack.apply(stream);
      }
      break;
    case QUDA_HALF_PRECISION:
      {
        PackFaceStaggered<short2,float> pack(buffer, &field, location, nFace, dagger, parity, dim, face_num, R, unpack);
        pack.apply(stream);
      }
      break;
    case QUDA_QUARTER_PRECISION:
      {
        PackFaceStaggered<char2,float> pack(buffer, &field, location, nFace, dagger, parity, dim, face_num, R, unpack);
        pack.apply(stream);  
      }
      break;
    default:
      errorQuda("Precision %d not supported", field.Precision());
    } // switch(field.Precision())
  }

#ifdef GPU_DOMAIN_WALL_DIRAC
  template <int dagger, typename FloatN>
    __global__ void packFaceDWKernel(PackParam<FloatN> param)
  {
    const int nFace = 1; // 1 face for dwf

#ifdef STRIPED
    const int sites_per_block = param.sites_per_block;
    int local_tid = threadIdx.x;
    int tid = sites_per_block * blockIdx.x + local_tid;
#else
    int tid = block_idx(param.swizzle) * blockDim.x + threadIdx.x;
    constexpr int sites_per_block = 1;
    constexpr int local_tid = 0;
#endif

    while ( local_tid < sites_per_block && tid < param.threads ) {

      // determine which dimension we are packing
      int face_idx;
      const int dim = dimFromFaceIndex(face_idx, tid, param);

      const int Ls = param.dc.X[4];

      // compute where the output is located
      // compute an index into the local volume from the index into the face
      // read spinor, spin-project, and write half spinor to face
      if (dim == 0) {
        // face_num determines which end of the lattice we are packing: 0 = beginning, 1 = end
        // FIXME these param.dc.ghostFaceCB constants do not incude the Ls dimension
        const int face_num = (face_idx >= nFace * Ls * param.dc.ghostFaceCB[0]) ? 1 : 0;
        face_idx -= face_num * nFace * Ls * param.dc.ghostFaceCB[0];
        if (face_num == 0) {
          const int idx = indexFromFaceIndex<5,QUDA_5D_PC,0,nFace,0>(face_idx,param);
          packFaceWilsonCore<0, dagger, 0>(param.out[0], param.outNorm[0], param.in, param.inNorm, idx, face_idx,
              Ls * param.dc.ghostFaceCB[0], param);
        } else {
          const int idx = indexFromFaceIndex<5,QUDA_5D_PC,0,nFace,1>(face_idx,param);
          packFaceWilsonCore<0, dagger, 1>(param.out[1], param.outNorm[1], param.in, param.inNorm, idx, face_idx,
              Ls * param.dc.ghostFaceCB[0], param);
        }
      } else if (dim == 1) {
        const int face_num = (face_idx >= nFace * Ls * param.dc.ghostFaceCB[1]) ? 1 : 0;
        face_idx -= face_num * nFace * Ls * param.dc.ghostFaceCB[1];
        if (face_num == 0) {
          const int idx = indexFromFaceIndex<5,QUDA_5D_PC,1,nFace,0>(face_idx,param);
          packFaceWilsonCore<1, dagger, 0>(param.out[2], param.outNorm[2], param.in, param.inNorm, idx, face_idx,
              Ls * param.dc.ghostFaceCB[1], param);
        } else {
          const int idx = indexFromFaceIndex<5,QUDA_5D_PC,1,nFace,1>(face_idx,param);
          packFaceWilsonCore<1, dagger, 1>(param.out[3], param.outNorm[3], param.in, param.inNorm, idx, face_idx,
              Ls * param.dc.ghostFaceCB[1], param);
        }
      } else if (dim == 2) {
        const int face_num = (face_idx >= nFace * Ls * param.dc.ghostFaceCB[2]) ? 1 : 0;
        face_idx -= face_num * nFace * Ls * param.dc.ghostFaceCB[2];
        if (face_num == 0) {
          const int idx = indexFromFaceIndex<5,QUDA_5D_PC,2,nFace,0>(face_idx,param);
          packFaceWilsonCore<2, dagger, 0>(param.out[4], param.outNorm[4], param.in, param.inNorm, idx, face_idx,
              Ls * param.dc.ghostFaceCB[2], param);
        } else {
          const int idx = indexFromFaceIndex<5,QUDA_5D_PC,2,nFace,1>(face_idx,param);
          packFaceWilsonCore<2, dagger, 1>(param.out[5], param.outNorm[5], param.in, param.inNorm, idx, face_idx,
              Ls * param.dc.ghostFaceCB[2], param);
        }
      } else {
        const int face_num = (face_idx >= nFace * Ls * param.dc.ghostFaceCB[3]) ? 1 : 0;
        face_idx -= face_num * nFace * Ls * param.dc.ghostFaceCB[3];
        if (face_num == 0) {
          const int idx = indexFromFaceIndex<5,QUDA_5D_PC,3,nFace,0>(face_idx,param);
          packFaceWilsonCore<3, dagger, 0>(param.out[6], param.outNorm[6], param.in, param.inNorm, idx, face_idx,
              Ls * param.dc.ghostFaceCB[3], param);
        } else {
          const int idx = indexFromFaceIndex<5,QUDA_5D_PC,3,nFace,1>(face_idx,param);
          packFaceWilsonCore<3, dagger, 1>(param.out[7], param.outNorm[7], param.in, param.inNorm, idx, face_idx,
              Ls * param.dc.ghostFaceCB[3], param);
        }
      }

#ifdef STRIPED
      local_tid += blockDim.x;
      tid += blockDim.x;
#else
      tid += blockDim.x*gridDim.x;
#endif
    } // while tid

  }


  template <int dagger, typename FloatN>
    __global__ void packFaceDW4DKernel(PackParam<FloatN> param)
  {
    const int nFace = 1; // 1 face for Wilson

#ifdef STRIPED
    const int sites_per_block = param.sites_per_block;
    int local_tid = threadIdx.x;
    int tid = sites_per_block * blockIdx.x + local_tid;
#else
    int tid = block_idx(param.swizzle) * blockDim.x + threadIdx.x;
    constexpr int sites_per_block = 1;
    constexpr int local_tid = 0;
#endif

    while ( local_tid < sites_per_block && tid < param.threads ) {

      const int Ls = param.dc.X[4];

      // determine which dimension we are packing
      int face_idx;
      const int dim = dimFromFaceIndex(face_idx, tid, param);

      // compute where the output is located
      // compute an index into the local volume from the index into the face
      // read spinor, spin-project, and write half spinor to face
      if (dim == 0) {
        // face_num determines which end of the lattice we are packing: 0 = beginning, 1 = end
        // FIXME these param.dc.ghostFaceCB constants do not incude the Ls dimension
        const int face_num = (face_idx >= nFace * Ls * param.dc.ghostFaceCB[0]) ? 1 : 0;
        face_idx -= face_num * nFace * Ls * param.dc.ghostFaceCB[0];
        if (face_num == 0) {
          const int idx = indexFromFaceIndex<5,QUDA_4D_PC,0,nFace,0>(face_idx,param);
          packFaceWilsonCore<0, dagger, 0>(param.out[0], param.outNorm[0], param.in, param.inNorm, idx, face_idx,
              Ls * param.dc.ghostFaceCB[0], param);
        } else {
          const int idx = indexFromFaceIndex<5,QUDA_4D_PC,0,nFace,1>(face_idx,param);
          packFaceWilsonCore<0, dagger, 1>(param.out[1], param.outNorm[1], param.in, param.inNorm, idx, face_idx,
              Ls * param.dc.ghostFaceCB[0], param);
        }
      } else if (dim == 1) {
        const int face_num = (face_idx >= nFace * Ls * param.dc.ghostFaceCB[1]) ? 1 : 0;
        face_idx -= face_num * nFace * Ls * param.dc.ghostFaceCB[1];
        if (face_num == 0) {
          const int idx = indexFromFaceIndex<5,QUDA_4D_PC,1,nFace,0>(face_idx,param);
          packFaceWilsonCore<1, dagger, 0>(param.out[2], param.outNorm[2], param.in, param.inNorm, idx, face_idx,
              Ls * param.dc.ghostFaceCB[1], param);
        } else {
          const int idx = indexFromFaceIndex<5,QUDA_4D_PC,1,nFace,1>(face_idx,param);
          packFaceWilsonCore<1, dagger, 1>(param.out[3], param.outNorm[3], param.in, param.inNorm, idx, face_idx,
              Ls * param.dc.ghostFaceCB[1], param);
        }
      } else if (dim == 2) {
        const int face_num = (face_idx >= nFace * Ls * param.dc.ghostFaceCB[2]) ? 1 : 0;
        face_idx -= face_num * nFace * Ls * param.dc.ghostFaceCB[2];
        if (face_num == 0) {
          const int idx = indexFromFaceIndex<5,QUDA_4D_PC,2,nFace,0>(face_idx,param);
          packFaceWilsonCore<2, dagger, 0>(param.out[4], param.outNorm[4], param.in, param.inNorm, idx, face_idx,
              Ls * param.dc.ghostFaceCB[2], param);
        } else {
          const int idx = indexFromFaceIndex<5,QUDA_4D_PC,2,nFace,1>(face_idx,param);
          packFaceWilsonCore<2, dagger, 1>(param.out[5], param.outNorm[5], param.in, param.inNorm, idx, face_idx,
              Ls * param.dc.ghostFaceCB[2], param);
        }
      } else {
        const int face_num = (face_idx >= nFace * Ls * param.dc.ghostFaceCB[3]) ? 1 : 0;
        face_idx -= face_num * nFace * Ls * param.dc.ghostFaceCB[3];
        if (face_num == 0) {
          const int idx = indexFromFaceIndex<5,QUDA_4D_PC,3,nFace,0>(face_idx,param);
          packFaceWilsonCore<3, dagger, 0>(param.out[6], param.outNorm[6], param.in, param.inNorm, idx, face_idx,
              Ls * param.dc.ghostFaceCB[3], param);
        } else {
          const int idx = indexFromFaceIndex<5,QUDA_4D_PC,3,nFace,1>(face_idx,param);
          packFaceWilsonCore<3, dagger, 1>(param.out[7], param.outNorm[7], param.in, param.inNorm, idx, face_idx,
              Ls * param.dc.ghostFaceCB[3], param);
        }
      }

#ifdef STRIPED
      local_tid += blockDim.x;
      tid += blockDim.x;
#else
      tid += blockDim.x*gridDim.x;
#endif
    } // while tid

  }

#endif

  template <typename FloatN, typename Float>
    class PackFaceDW : public PackFace<FloatN, Float> {

  private:

    int inputPerSite() const { return 24; } // input is full spinor
    int outputPerSite() const { return 12; } // output is spin projected

  public:
    PackFaceDW(void *faces[], const cudaColorSpinorField *in, MemoryLocation location,
	       const int dagger, const int parity)
    : PackFace<FloatN, Float>(faces, in, location, dagger, parity, 1) { }
    virtual ~PackFaceDW() { }

    void apply(const cudaStream_t &stream) {
      TuneParam tp = tuneLaunch(*this, getTuning(), getVerbosity());

#ifdef GPU_DOMAIN_WALL_DIRAC
      static PackParam<FloatN> param;
      this->prepareParam(param,tp);
      void *args[] = { &param };
      void (*func)(PackParam<FloatN>) = this->dagger ? &(packFaceDWKernel<1,FloatN>) : &(packFaceDWKernel<0,FloatN>);
      cudaLaunchKernel( (const void*)func, tp.grid, tp.block, args, tp.shared_bytes, stream);
#else
      errorQuda("DW face packing kernel is not built");
#endif
    }

    long long flops() const { return outputPerSite()*this->threads(); }
  };

  template <typename FloatN, typename Float>
    class PackFaceDW4D : public PackFace<FloatN, Float> {

  private:

    int inputPerSite() const { return 24; } // input is full spinor
    int outputPerSite() const { return 12; } // output is spin projected

  public:
    PackFaceDW4D(void *faces[], const cudaColorSpinorField *in, MemoryLocation location,
		 const int dagger, const int parity)
      : PackFace<FloatN, Float>(faces, in, location, dagger, parity, 1) { }
    virtual ~PackFaceDW4D() { }

    void apply(const cudaStream_t &stream) {
      TuneParam tp = tuneLaunch(*this, getTuning(), getVerbosity());

#ifdef GPU_DOMAIN_WALL_DIRAC
      static PackParam<FloatN> param;
      this->prepareParam(param,tp);
      void *args[] = { &param };
      void (*func)(PackParam<FloatN>) = this->dagger ? &(packFaceDW4DKernel<1,FloatN>) : &(packFaceDW4DKernel<0,FloatN>);
      cudaLaunchKernel( (const void*)func, tp.grid, tp.block, args, tp.shared_bytes, stream);
#else
      errorQuda("4D preconditioned DW face packing kernel is not built");
#endif
    }

    long long flops() const { return outputPerSite()*this->threads(); }
  };

  void packFaceDW(void *ghost_buf[], cudaColorSpinorField &in, MemoryLocation location, const int dagger,
		  const int parity, const cudaStream_t &stream) {

    if (in.PCType() == QUDA_4D_PC) {
      switch (in.Precision()) {
      case QUDA_DOUBLE_PRECISION: {
        PackFaceDW4D<double2, double> pack(ghost_buf, &in, location, dagger, parity);
        pack.apply(stream);
      } break;
      case QUDA_SINGLE_PRECISION: {
        PackFaceDW4D<float4, float> pack(ghost_buf, &in, location, dagger, parity);
        pack.apply(stream);
      } break;
      case QUDA_HALF_PRECISION: {
        PackFaceDW4D<short4, float> pack(ghost_buf, &in, location, dagger, parity);
        pack.apply(stream);
      } break;
      case QUDA_QUARTER_PRECISION: {
        PackFaceDW4D<char4, float> pack(ghost_buf, &in, location, dagger, parity);
        pack.apply(stream);
      } break;
      default: errorQuda("Precision %d not supported", in.Precision());
      }
    } else {
      switch (in.Precision()) {
      case QUDA_DOUBLE_PRECISION: {
        PackFaceDW<double2, double> pack(ghost_buf, &in, location, dagger, parity);
        pack.apply(stream);
      } break;
      case QUDA_SINGLE_PRECISION: {
        PackFaceDW<float4, float> pack(ghost_buf, &in, location, dagger, parity);
        pack.apply(stream);
      } break;
      case QUDA_HALF_PRECISION: {
        PackFaceDW<short4, float> pack(ghost_buf, &in, location, dagger, parity);
        pack.apply(stream);
      } break;
      case QUDA_QUARTER_PRECISION: {
        PackFaceDW<char4, float> pack(ghost_buf, &in, location, dagger, parity);
        pack.apply(stream);
      } break;
      default: errorQuda("Precision %d not supported", in.Precision());
      }
    }
  }

#ifdef GPU_NDEG_TWISTED_MASS_DIRAC
  template <int dagger, typename FloatN>
    __global__ void packFaceNdegTMKernel(PackParam<FloatN> param)
  {
    const int nFace = 1; // 1 face for Wilson
    const int Nf = 2;

#ifdef STRIPED
    const int sites_per_block = param.sites_per_block;
    int local_tid = threadIdx.x;
    int tid = sites_per_block * blockIdx.x + local_tid;
#else
    int tid = block_idx(param.swizzle) * blockDim.x + threadIdx.x;
    constexpr int sites_per_block = 1;
    constexpr int local_tid = 0;
#endif

    while ( local_tid < sites_per_block && tid < param.threads ) {

      // determine which dimension we are packing
      int face_idx;
      const int dim = dimFromFaceIndex(face_idx, tid, param);

      // compute where the output is located
      // compute an index into the local volume from the index into the face
      // read spinor, spin-project, and write half spinor to face
      if (dim == 0) {
        // face_num determines which end of the lattice we are packing:
        // 0 = beginning, 1 = end FIXME these param.dc.ghostFaceCB constants
        // do not include the Nf dimension
        const int face_num = (face_idx >= nFace * Nf * param.dc.ghostFaceCB[0]) ? 1 : 0;
        face_idx -= face_num * nFace * Nf * param.dc.ghostFaceCB[0];
        if (face_num == 0) {
          const int idx = indexFromFaceIndex<5,QUDA_4D_PC,0,nFace,0>(face_idx,param);
          packFaceWilsonCore<0, dagger, 0>(param.out[0], param.outNorm[0], param.in, param.inNorm, idx, face_idx,
              Nf * param.dc.ghostFaceCB[0], param);
        } else {
          const int idx = indexFromFaceIndex<5,QUDA_4D_PC,0,nFace,1>(face_idx,param);
          packFaceWilsonCore<0, dagger, 1>(param.out[1], param.outNorm[1], param.in, param.inNorm, idx, face_idx,
              Nf * param.dc.ghostFaceCB[0], param);
        }
      } else if (dim == 1) {
        const int face_num = (face_idx >= nFace * Nf * param.dc.ghostFaceCB[1]) ? 1 : 0;
        face_idx -= face_num * nFace * Nf * param.dc.ghostFaceCB[1];
        if (face_num == 0) {
          const int idx = indexFromFaceIndex<5,QUDA_4D_PC,1,nFace,0>(face_idx,param);
          packFaceWilsonCore<1, dagger, 0>(param.out[2], param.outNorm[2], param.in, param.inNorm, idx, face_idx,
              Nf * param.dc.ghostFaceCB[1], param);
        } else {
          const int idx = indexFromFaceIndex<5,QUDA_4D_PC,1,nFace,1>(face_idx,param);
          packFaceWilsonCore<1, dagger, 1>(param.out[3], param.outNorm[3], param.in, param.inNorm, idx, face_idx,
              Nf * param.dc.ghostFaceCB[1], param);
        }
      } else if (dim == 2) {
        const int face_num = (face_idx >= nFace * Nf * param.dc.ghostFaceCB[2]) ? 1 : 0;
        face_idx -= face_num * nFace * Nf * param.dc.ghostFaceCB[2];
        if (face_num == 0) {
          const int idx = indexFromFaceIndex<5,QUDA_4D_PC,2,nFace,0>(face_idx,param);
          packFaceWilsonCore<2, dagger, 0>(param.out[4], param.outNorm[4], param.in, param.inNorm, idx, face_idx,
              Nf * param.dc.ghostFaceCB[2], param);
        } else {
          const int idx = indexFromFaceIndex<5,QUDA_4D_PC,2,nFace,1>(face_idx,param);
          packFaceWilsonCore<2, dagger, 1>(param.out[5], param.outNorm[5], param.in, param.inNorm, idx, face_idx,
              Nf * param.dc.ghostFaceCB[2], param);
        }
      } else {
        const int face_num = (face_idx >= nFace * Nf * param.dc.ghostFaceCB[3]) ? 1 : 0;
        face_idx -= face_num * nFace * Nf * param.dc.ghostFaceCB[3];
        if (face_num == 0) {
          const int idx = indexFromFaceIndex<5,QUDA_4D_PC,3,nFace,0>(face_idx,param);
          packFaceWilsonCore<3, dagger, 0>(param.out[6], param.outNorm[6], param.in, param.inNorm, idx, face_idx,
              Nf * param.dc.ghostFaceCB[3], param);
        } else {
          const int idx = indexFromFaceIndex<5,QUDA_4D_PC,3,nFace,1>(face_idx,param);
          packFaceWilsonCore<3, dagger, 1>(param.out[7], param.outNorm[7], param.in, param.inNorm, idx, face_idx,
              Nf * param.dc.ghostFaceCB[3], param);
        }
      }

#ifdef STRIPED
      local_tid += blockDim.x;
      tid += blockDim.x;
#else
      tid += blockDim.x*gridDim.x;
#endif
    } // while tid

  }

#endif

  template <typename FloatN, typename Float>
    class PackFaceNdegTM : public PackFace<FloatN, Float> {

  private:

    int inputPerSite() const { return 24; } // input is full spinor
    int outputPerSite() const { return 12; } // output is spin projected

  public:
    PackFaceNdegTM(void *faces[], const cudaColorSpinorField *in, MemoryLocation location,
		   const int dagger, const int parity)
      : PackFace<FloatN, Float>(faces, in, location, dagger, parity, 1) { }
    virtual ~PackFaceNdegTM() { }

    void apply(const cudaStream_t &stream) {
      TuneParam tp = tuneLaunch(*this, getTuning(), getVerbosity());

#ifdef GPU_NDEG_TWISTED_MASS_DIRAC
      static PackParam<FloatN> param;
      this->prepareParam(param,tp);
      void *args[] = { &param };
      void (*func)(PackParam<FloatN>) = this->dagger ? &(packFaceNdegTMKernel<1,FloatN>) : &(packFaceNdegTMKernel<0,FloatN>);
      cudaLaunchKernel( (const void*)func, tp.grid, tp.block, args, tp.shared_bytes, stream);
#else
      errorQuda("Non-degenerate twisted mass face packing kernel is not built");
#endif
    }

    long long flops() const { return outputPerSite()*this->threads(); }
  };

  void packFaceNdegTM(void *ghost_buf[], cudaColorSpinorField &in, MemoryLocation location, const int dagger,
		      const int parity, const cudaStream_t &stream) {

    switch(in.Precision()) {
    case QUDA_DOUBLE_PRECISION:
      {
        PackFaceNdegTM<double2, double> pack(ghost_buf, &in, location, dagger, parity);
        pack.apply(stream);
      }
      break;
    case QUDA_SINGLE_PRECISION:
      {
        PackFaceNdegTM<float4, float> pack(ghost_buf, &in, location, dagger, parity);
        pack.apply(stream);
      }
      break;
    case QUDA_HALF_PRECISION:
      {
        PackFaceNdegTM<short4, float> pack(ghost_buf, &in, location, dagger, parity);
        pack.apply(stream);
      }
      break;
    case QUDA_QUARTER_PRECISION:
      {
        PackFaceNdegTM<char4, float> pack(ghost_buf, &in, location, dagger, parity);
        pack.apply(stream);
      }
      break;
    default:
      errorQuda("Precision %d not supported", in.Precision());
    }
  }

  void packFace(void *ghost_buf[2*QUDA_MAX_DIM], cudaColorSpinorField &in,
		MemoryLocation location, const int nFace,
		const int dagger, const int parity,
		const int dim, const int face_num,
		const cudaStream_t &stream,
		const double a, const double b)
  {
    int nDimPack = 0;
    if(dim < 0){
      for (int d=0; d<4; d++) {
	if(!commDim[d]) continue;
	if (d != 3 || getKernelPackT() || a != 0.0 || b!= 0.0) nDimPack++;
      }
    }else{
      if(commDim[dim]){
	if(dim!=3 || getKernelPackT() || a!=0.0 || b != 0.0) nDimPack++;
      }
    }
    if (!nDimPack) return; // if zero then we have nothing to pack

    if (nFace != 1 && in.Nspin() != 1)
      errorQuda("Unsupported number of faces %d", nFace);

    // Need to update this logic for other multi-src dslash packing
    if (in.Nspin() == 1) {
      packFaceStaggered(ghost_buf, in, location, nFace, dagger, parity, dim, face_num, stream);
    } else if (a!=0.0 || b!=0.0) {
      // Need to update this logic for other multi-src dslash packing
      if(in.TwistFlavor() == QUDA_TWIST_SINGLET) {
	packTwistedFaceWilson(ghost_buf, in, location, dagger, parity, a, b, stream);
      } else {
	errorQuda("Cannot perform twisted packing for the spinor.");
      }
    } else if (in.Ndim() == 5) {
      if(in.TwistFlavor() == QUDA_TWIST_INVALID) {
	packFaceDW(ghost_buf, in, location, dagger, parity, stream);
      } else {
	packFaceNdegTM(ghost_buf, in, location, dagger, parity, stream);
      }
    } else {
      packFaceWilson(ghost_buf, in, location, dagger, parity, stream);
    }
  }



  void packFaceExtended(void* buffer[2*QUDA_MAX_DIM], cudaColorSpinorField &field,
			MemoryLocation location, const int nFace, const int R[],
			const int dagger, const int parity, const int dim, const int face_num,
			const cudaStream_t &stream, const bool unpack)
  {
    int nDimPack = 0;
    if(dim < 0){
      for(int d=0; d<4; d++){
	if(R[d]) nDimPack++;
      }
    }else{
      if(R[dim]) nDimPack++;
    }

    if(!nDimPack) return; // if zero then we have nothing to pack
    if(field.Nspin() == 1){
      packFaceExtendedStaggered(buffer, field, location, nFace, R, dagger, parity, dim, face_num, stream, unpack);
    }else{
      errorQuda("Extended quark field is not supported");
    }

  }

#endif // MULTI_GPU

} // namespace quda

#endif
