
#include <hip/hip_runtime.h>
#ifdef USE_LEGACY_DSLASH

#include <cstdlib>
#include <cstdio>
#include <string>
#include <iostream>
#include <typeinfo>

#include <color_spinor_field.h>
#include <clover_field.h>

// these control the Wilson-type actions
#ifdef GPU_WILSON_DIRAC
//#define DIRECT_ACCESS_LINK
//#define DIRECT_ACCESS_WILSON_SPINOR
//#define DIRECT_ACCESS_WILSON_ACCUM
//#define DIRECT_ACCESS_WILSON_INTER
//#define DIRECT_ACCESS_WILSON_PACK_SPINOR
//#define DIRECT_ACCESS_CLOVER
#endif // GPU_WILSON_DIRAC

#include <quda_internal.h>
#include <dslash_quda.h>
#include <dslash.h>
#include <sys/time.h>
#include <blas_quda.h>

#include <inline_ptx.h>
#include <dslash_policy.cuh>

namespace quda {

  namespace mobius {

#undef GPU_STAGGERED_DIRAC
#include <dslash_constants.h>
#include <dslash_textures.h>
#include <dslash_index.cuh>

    // Enable shared memory dslash for Fermi architecture
    //#define SHARED_WILSON_DSLASH
    //#define SHARED_8_BYTE_WORD_SIZE // 8-byte shared memory access

#ifdef GPU_DOMAIN_WALL_DIRAC
#include <mdw_dslash4_def.h>      // Dslash4, intermediate operator for Mobius Mat_4 kernels
#include <mdw_dslash4pre_def.h>   // Dslash4pre, intermediate operator for Mobius Mat_4 kernels
#include <mdw_dslash5_def.h>      // Dslash5 Mobius Domain Wall kernels
#include <mdw_dslash5inv_def.h>   // Dslash5inv Mobius Domain Wall kernels
#endif

#ifndef DSLASH_SHARED_FLOATS_PER_THREAD
#define DSLASH_SHARED_FLOATS_PER_THREAD 0
#endif

#include <dslash_quda.cuh>
  }

  using namespace mobius;

#ifdef GPU_DOMAIN_WALL_DIRAC
  //Dslash class definition for Mobius Domain Wall Fermion
  template <typename sFloat, typename gFloat>
  class MDWFDslashPCCuda : public DslashCuda {

  private:
    const int DS_type;

    bool checkGrid(TuneParam &param) const {
      if (param.grid.x > (unsigned int)deviceProp.maxGridSize[0] || param.grid.y > (unsigned int)deviceProp.maxGridSize[1]) {
        warningQuda("Autotuner is skipping blockDim=(%u,%u,%u), gridDim=(%u,%u,%u) because lattice volume is too large",
		    param.block.x, param.block.y, param.block.z, 
		    param.grid.x, param.grid.y, param.grid.z);
        return false;
      } else {
        return true;
      }
    }

  protected:
    bool advanceBlockDim(TuneParam &param) const
    {
      const unsigned int max_shared = 16384; // FIXME: use deviceProp.sharedMemPerBlock;
      const int step[2] = { deviceProp.warpSize, 1 };
      bool advance[2] = { false, false };

      // first try to advance block.x
      param.block.x += step[0];
      //memory constraint
      if (param.block.x > (unsigned int)deviceProp.maxThreadsDim[0] ||
          sharedBytesPerThread()*param.block.x*param.block.y > max_shared) {
        advance[0] = false;
        param.block.x = step[0]; // reset block.x
      } else {
        advance[0] = true; // successfully advanced block.x
      }

      if (!advance[0]) {  // if failed to advance block.x, now try block.y
        param.block.y += step[1];

	//memory constraint
        if (param.block.y > (unsigned)in->X(4) ||
            sharedBytesPerThread()*param.block.x*param.block.y > max_shared) {
          advance[1] = false;
          param.block.y = step[1]; // reset block.y
        } else {
          advance[1] = true; // successfully advanced block.y
        }
      }

      //thread constraint
      if ( (advance[0] || advance[1]) && param.block.x*param.block.y*param.block.z <= (unsigned)deviceProp.maxThreadsPerBlock) {
        param.grid = dim3( (dslashParam.threads+param.block.x-1) / param.block.x, 
			   (in->X(4)+param.block.y-1) / param.block.y, 1);

        bool advance = true;
        if (!checkGrid(param)) advance = advanceBlockDim(param);
        return advance;
      } else {
        return false;
      }
    }

    unsigned int sharedBytesPerThread() const { return 0; }
  
  public:
    MDWFDslashPCCuda(cudaColorSpinorField *out, const GaugeField &gauge, const cudaColorSpinorField *in,
		     const cudaColorSpinorField *x, const double mferm, const double a,
                     const double *b_5, const double *c_5, const double m5,
                     const int parity, const int dagger, const int *commOverride, const int DS_type)
      : DslashCuda(out, in, x, gauge, parity, dagger, commOverride), DS_type(DS_type)
    { 
      dslashParam.a = a;
      dslashParam.a_f = a;
      dslashParam.mferm = mferm;
      dslashParam.mferm_f = mferm;

      memcpy(dslashParam.mdwf_b5_d, b_5, out->X(4)*sizeof(double));
      memcpy(dslashParam.mdwf_c5_d, c_5, out->X(4)*sizeof(double));
      for (int s=0; s<out->X(4); s++) {
        dslashParam.mdwf_b5_f[s] = (float)dslashParam.mdwf_b5_d[s];
        dslashParam.mdwf_c5_f[s] = (float)dslashParam.mdwf_c5_d[s];
      }

      dslashParam.m5_d = m5;
      dslashParam.m5_f = (float)m5;
    }
    virtual ~MDWFDslashPCCuda() { unbindSpinorTex<sFloat>(in, out, x); }

    TuneKey tuneKey() const
    {
      TuneKey key = DslashCuda::tuneKey();
      switch(DS_type){
      case 0:
	strcat(key.aux,",Dslash4");
	break;
      case 1:
	strcat(key.aux,",Dslash4pre");
	break;
      case 2:
	strcat(key.aux,",Dslash5");
	break;
      case 3:
	strcat(key.aux,",Dslash5inv");
	break;
      }
      return key;
    }

    virtual void initTuneParam(TuneParam &param) const
    {
      Tunable::initTuneParam(param);
      param.grid = dim3( (dslashParam.threads+param.block.x-1) / param.block.x, 
			 (in->X(4)+param.block.y-1) / param.block.y, 1);
      bool ok = true;
      if (!checkGrid(param)) ok = advanceBlockDim(param);
      if (!ok) errorQuda("Lattice volume is too large for even the largest blockDim");
    }

    /** sets default values for when tuning is disabled */
    virtual void defaultTuneParam(TuneParam &param) const
    {
      Tunable::defaultTuneParam(param);
      param.grid = dim3( (dslashParam.threads+param.block.x-1) / param.block.x, 
			 (in->X(4)+param.block.y-1) / param.block.y, 1);
      bool ok = true;
      if (!checkGrid(param)) ok = advanceBlockDim(param);
      if (!ok) errorQuda("Lattice volume is too large for even the largest blockDim");
    }

    void apply(const cudaStream_t &stream)
    {
#ifndef USE_TEXTURE_OBJECTS
      if (dslashParam.kernel_type == INTERIOR_KERNEL) bindSpinorTex<sFloat>(in, out, x);
#endif // USE_TEXTURE_OBJECTS
      TuneParam tp = tuneLaunch(*this, getTuning(), getVerbosity());
      setParam();
      switch(DS_type){
      case 0:
	DSLASH(MDWFDslash4, tp.grid, tp.block, tp.shared_bytes, stream, dslashParam);
	break;
      case 1:
	DSLASH(MDWFDslash4pre, tp.grid, tp.block, tp.shared_bytes, stream, dslashParam);
	break;
      case 2:
	DSLASH(MDWFDslash5, tp.grid, tp.block, tp.shared_bytes, stream, dslashParam);
	break;
      case 3:
	DSLASH(MDWFDslash5inv, tp.grid, tp.block, tp.shared_bytes, stream, dslashParam);
	break;
      default:
	errorQuda("invalid Dslash type");
      }
    }

    long long flops() const {
      long long Ls = in->X(4);
      long long vol4d = in->VolumeCB() / Ls;
      long long bulk = (Ls-2)*vol4d;
      long long wall = 2*vol4d;
      long long flops = 0;
      switch(DS_type){
      case 0:
	flops = DslashCuda::flops();
	break;
      case 1:
	flops = 72ll*in->VolumeCB() + 96ll*bulk + 120ll*wall;
	break;
      case 2:
	flops = (x ? 96ll : 48ll)*in->VolumeCB() + 96ll*bulk + 120ll*wall;
	break;
      case 3:
	flops = 144ll*in->VolumeCB()*Ls + 3ll*Ls*(Ls-1ll);
	break;
      default:
	errorQuda("invalid Dslash type");
      }
      return flops;
    }

    long long bytes() const {
      bool isFixed = (in->Precision() == sizeof(short) || in->Precision() == sizeof(char)) ? true : false;
      int spinor_bytes = 2 * in->Ncolor() * in->Nspin() * in->Precision() + (isFixed ? sizeof(float) : 0);
      long long Ls = in->X(4);
      long long bytes = 0;

      switch(DS_type){
      case 0:
	bytes = DslashCuda::bytes();
	break;
      case 1:
      case 2:
	bytes = (x ? 5ll : 4ll) * spinor_bytes * in->VolumeCB();
	break;
      case 3:
	bytes = (x ? Ls + 2 : Ls + 1) * spinor_bytes * in->VolumeCB();
	break;
      default:
	errorQuda("invalid Dslash type");
      }
      return bytes;
    }
  };
#endif // GPU_DOMAIN_WALL_DIRAC

  //-----------------------------------------------------
  // Modification for 4D preconditioned Mobius DWF operator
  // Additional Arg. is added to give a function name.
  //
  // pre-defined DS_type list
  // 0 = MDWF dslash4
  // 1 = MDWF dslash4pre
  // 2 = MDWF dslash5
  // 3 = MDWF dslash5inv
  //-----------------------------------------------------

  void MDWFDslashCuda(cudaColorSpinorField *out, const cudaGaugeField &gauge,
		      const cudaColorSpinorField *in, const int parity, const int dagger,
		      const cudaColorSpinorField *x, const double &m_f, const double &k2,
                      const double *b_5, const double *c_5, const double &m5,
		      const int *commOverride, const int DS_type, TimeProfile &profile)
  {
#ifdef GPU_DOMAIN_WALL_DIRAC
    using namespace dslash;
    const_cast<cudaColorSpinorField*>(in)->createComms(1);

    DslashCuda *dslash = nullptr;
    if (in->Precision() == QUDA_DOUBLE_PRECISION) {
      dslash = new MDWFDslashPCCuda<double2,double2>(out, gauge, in, x, m_f, k2, b_5, c_5, m5, parity, dagger, commOverride, DS_type);
    } else if (in->Precision() == QUDA_SINGLE_PRECISION) {
      dslash = new MDWFDslashPCCuda<float4,float4>(out, gauge, in, x, m_f, k2, b_5, c_5, m5, parity, dagger, commOverride, DS_type);
    } else if (in->Precision() == QUDA_HALF_PRECISION) {
      dslash = new MDWFDslashPCCuda<short4,short4>(out, gauge, in, x, m_f, k2, b_5, c_5, m5, parity, dagger, commOverride, DS_type);
    }

    // the parameters passed to dslashCuda must be 4-d volume and 3-d
    // faces because Ls is added as the y-dimension in thread space
    int ghostFace[QUDA_MAX_DIM];
    for (int i=0; i<4; i++) ghostFace[i] = in->GhostFace()[i] / in->X(4);

    DslashPolicyImp<DslashCuda> *dslashImp = nullptr;
    if (DS_type != 0) {
      dslashImp = DslashFactory<DslashCuda>::create(QudaDslashPolicy::QUDA_DSLASH_NC);
      (*dslashImp)(*dslash, const_cast<cudaColorSpinorField*>(in), in->Volume()/in->X(4), ghostFace, profile);
      delete dslashImp;
    } else {
      DslashPolicyTune<DslashCuda> dslash_policy(
          *dslash, const_cast<cudaColorSpinorField *>(in), in->Volume() / in->X(4), ghostFace, profile);
      dslash_policy.apply(0);
    }

    delete dslash;
#else
    errorQuda("Domain wall dslash has not been built");
#endif
  }

}

#endif
