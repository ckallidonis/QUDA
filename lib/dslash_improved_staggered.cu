#include "hip/hip_runtime.h"
#include <cstdlib>
#include <cstdio>
#include <string>
#include <iostream>
#include <typeinfo>

#include <color_spinor_field.h>
#include <clover_field.h>

//these are access control for staggered action
#if (defined GPU_STAGGERED_DIRAC && defined USE_LEGACY_DSLASH)
#if (__COMPUTE_CAPABILITY__ >= 300) // Kepler works best with texture loads only
//#define DIRECT_ACCESS_FAT_LINK
//#define DIRECT_ACCESS_LONG_LINK
//#define DIRECT_ACCESS_SPINOR
//#define DIRECT_ACCESS_ACCUM
//#define DIRECT_ACCESS_INTER
//#define DIRECT_ACCESS_PACK
#else // Fermi
//#define DIRECT_ACCESS_FAT_LINK
//#define DIRECT_ACCESS_LONG_LINK
//#define DIRECT_ACCESS_SPINOR
//#define DIRECT_ACCESS_ACCUM
//#define DIRECT_ACCESS_INTER
//#define DIRECT_ACCESS_PACK
#endif
#endif // GPU_STAGGERED_DIRAC

#include <quda_internal.h>
#include <dslash_quda.h>
#include <dslash.h>
#include <sys/time.h>
#include <blas_quda.h>

#include <inline_ptx.h>
#include <dslash_policy.cuh>

namespace quda {
#ifdef USE_LEGACY_DSLASH
  namespace improvedstaggered {
#include <dslash_constants.h>
#include <dslash_textures.h>
#include <dslash_index.cuh>

#undef GPU_NDEG_TWISTED_MASS_DIRAC
#undef GPU_CLOVER_DIRAC
#undef GPU_DOMAIN_WALL_DIRAC
#define DD_IMPROVED 1

#define DD_DAG 0
#include <staggered_dslash_def.h> // staggered Dslash kernels
#undef DD_DAG
#define DD_DAG 1
#include <staggered_dslash_def.h> // staggered Dslash dagger kernels

#undef DD_IMPROVED

#include <dslash_quda.cuh>
  } // end namespace improvedstaggered
#endif

  using namespace improvedstaggered;

#if (defined GPU_STAGGERED_DIRAC && defined USE_LEGACY_DSLASH)
  template <typename sFloat, typename fatGFloat, typename longGFloat, typename phaseFloat>
  class StaggeredDslashCuda : public DslashCuda {

  private:
    const GaugeField &fatGauge;
    const GaugeField &longGauge;
    const unsigned int nSrc;

  protected:
    bool tuneAuxDim() const { return true; } // Do tune the aux dimensions.
    unsigned int sharedBytesPerThread() const
    {
#ifdef PARALLEL_DIR
      int reg_size = (typeid(sFloat)==typeid(double2) ? sizeof(double) : sizeof(float));
      return 6 * reg_size;
#else
      return 0;
#endif
    }

  public:
    StaggeredDslashCuda(cudaColorSpinorField *out, const GaugeField &fatGauge, const GaugeField &longGauge,
                        const cudaColorSpinorField *in, const cudaColorSpinorField *x, const double a,
                        const int parity, const int dagger, const int *commOverride)
      : DslashCuda(out, in, x, longGauge, parity, dagger, commOverride),
        fatGauge(fatGauge), longGauge(longGauge), nSrc(in->X(4))
    { 
#ifdef MULTI_GPU
      for(int i=0;i < 4; i++){
        if(comm_dim_partitioned(i) && (fatGauge.X()[i] < 6)){
          errorQuda("ERROR: partitioned dimension with local size less than 6 is not supported in improved staggered dslash\n");
        }
      }
#endif

      bindFatGaugeTex(static_cast<const cudaGaugeField&>(fatGauge), parity, dslashParam);
      bindLongGaugeTex(static_cast<const cudaGaugeField&>(longGauge), parity, dslashParam);

      if (in->Precision() != fatGauge.Precision() || in->Precision() != longGauge.Precision()){
        errorQuda("Mixing gauge and spinor precision not supported"
                  "(precision=%d, fatlinkGauge.precision=%d, longGauge.precision=%d",
                  in->Precision(), fatGauge.Precision(), longGauge.Precision());
      }

      dslashParam.a = a;
      dslashParam.a_f = a;
      dslashParam.fat_link_max = fatGauge.LinkMax();
      dslashParam.coeff = 1.0/longGauge.Scale();
      dslashParam.coeff_f = (float)dslashParam.coeff;
    }

    virtual ~StaggeredDslashCuda() {
      unbindSpinorTex<sFloat>(in, out, x);
      unbindFatGaugeTex(static_cast<const cudaGaugeField&>(fatGauge));
      unbindLongGaugeTex(static_cast<const cudaGaugeField&>(longGauge));
    }

    void apply(const hipStream_t &stream)
    {
#ifndef USE_TEXTURE_OBJECTS
      if (dslashParam.kernel_type == INTERIOR_KERNEL) bindSpinorTex<sFloat>(in, out, x);
#endif // USE_TEXTURE_OBJECTS
      TuneParam tp = tuneLaunch(*this, getTuning(), getVerbosity());
      setParam();
      dslashParam.gauge_stride = fatGauge.Stride();
      dslashParam.long_gauge_stride = longGauge.Stride();
      dslashParam.swizzle = tp.aux.x;
      IMPROVED_STAGGERED_DSLASH(tp.grid, tp.block, tp.shared_bytes, stream, dslashParam);
    }

    bool advanceBlockDim(TuneParam &param) const
    {
      const unsigned int max_shared = deviceProp.sharedMemPerBlock;
      // first try to advance block.y (number of right-hand sides per block)
      if (param.block.y < nSrc && param.block.y < (unsigned int)deviceProp.maxThreadsDim[1] &&
	  sharedBytesPerThread()*param.block.x*param.block.y < max_shared &&
	  (param.block.x*(param.block.y+1u)) <= (unsigned int)deviceProp.maxThreadsPerBlock) {
	param.block.y++;
	param.grid.y = (nSrc + param.block.y - 1) / param.block.y;
	return true;
      } else {
	bool rtn = DslashCuda::advanceBlockDim(param);
	param.block.y = 1;
	param.grid.y = nSrc;
	return rtn;
      }
    }

    bool advanceAux(TuneParam &param) const
    {
#ifdef SWIZZLE
      if (param.aux.x < 2*deviceProp.multiProcessorCount) {
        param.aux.x++;
	return true;
      } else {
        param.aux.x = 1;
	return false;
      }
#else
      return false;
#endif
    }

    void initTuneParam(TuneParam &param) const
    {
      DslashCuda::initTuneParam(param);
      param.block.y = 1;
      param.grid.y = nSrc;
      param.aux.x = 1;
    }

    void defaultTuneParam(TuneParam &param) const { initTuneParam(param); }

    int Nface() const { return 6; }

    /*
      per direction / dimension flops
      SU(3) matrix-vector flops = (8 Nc - 2) * Nc
      xpay = 2 * 2 * Nc * Ns
      
      So for the full dslash we have      
      flops = (2 * 2 * Nd * (8*Nc-2) * Nc)  +  ((2 * 2 * Nd - 1) * 2 * Nc * Ns)
      flops_xpay = flops + 2 * 2 * Nc * Ns
      
      For Asqtad this should give 1146 for Nc=3,Ns=2 and 1158 for the axpy equivalent
    */
    virtual long long flops() const {
      int mv_flops = (8 * in->Ncolor() - 2) * in->Ncolor(); // SU(3) matrix-vector flops
      int ghost_flops = (3 + 1) * (mv_flops + 2*in->Ncolor()*in->Nspin());
      int xpay_flops = 2 * 2 * in->Ncolor() * in->Nspin(); // multiply and add per real component
      int num_dir = 2 * 4; // dir * dim

      long long flops = 0;
      switch(dslashParam.kernel_type) {
      case EXTERIOR_KERNEL_X:
      case EXTERIOR_KERNEL_Y:
      case EXTERIOR_KERNEL_Z:
      case EXTERIOR_KERNEL_T:
	flops = ghost_flops * 2 * in->GhostFace()[dslashParam.kernel_type];
	break;
      case EXTERIOR_KERNEL_ALL:
	{
	  long long ghost_sites = 2 * (in->GhostFace()[0]+in->GhostFace()[1]+in->GhostFace()[2]+in->GhostFace()[3]);
	  flops = ghost_flops * ghost_sites;
	  break;
	}
      case INTERIOR_KERNEL:
      case KERNEL_POLICY:
	{
	  long long sites = in->VolumeCB();
	  flops = (2*num_dir*mv_flops +                   // SU(3) matrix-vector multiplies
		   (2*num_dir-1)*2*in->Ncolor()*in->Nspin()) * sites;   // accumulation
	  if (x) flops += xpay_flops * sites; // axpy is always on interior

	  if (dslashParam.kernel_type == KERNEL_POLICY) break;
	  // now correct for flops done by exterior kernel
	  long long ghost_sites = 0;
	  for (int d=0; d<4; d++) if (dslashParam.commDim[d]) ghost_sites += 2 * in->GhostFace()[d];
	  flops -= ghost_flops * ghost_sites;
	  
	  break;
	}
      }
      return flops;
    }

    virtual long long bytes() const {
      int gauge_bytes_fat = QUDA_RECONSTRUCT_NO * in->Precision();
      int gauge_bytes_long = reconstruct * in->Precision();
      bool isFixed = (in->Precision() == sizeof(short) || in->Precision() == sizeof(char)) ? true : false;
      int spinor_bytes = 2 * in->Ncolor() * in->Nspin() * in->Precision() + (isFixed ? sizeof(float) : 0);
      int ghost_bytes = 3 * (spinor_bytes + gauge_bytes_long) + (spinor_bytes + gauge_bytes_fat) + spinor_bytes;
      int num_dir = 2 * 4; // set to 4 dimensions since we take care of 5-d fermions in derived classes where necessary

      long long bytes = 0;
      switch(dslashParam.kernel_type) {
      case EXTERIOR_KERNEL_X:
      case EXTERIOR_KERNEL_Y:
      case EXTERIOR_KERNEL_Z:
      case EXTERIOR_KERNEL_T:
	bytes = ghost_bytes * 2 * in->GhostFace()[dslashParam.kernel_type];
	break;
      case EXTERIOR_KERNEL_ALL:
	{
	  long long ghost_sites = 2 * (in->GhostFace()[0]+in->GhostFace()[1]+in->GhostFace()[2]+in->GhostFace()[3]);
	  bytes = ghost_bytes * ghost_sites;
	  break;
	}
      case INTERIOR_KERNEL:
      case KERNEL_POLICY:
	{
	  long long sites = in->VolumeCB();
	  bytes = (num_dir*(gauge_bytes_fat + gauge_bytes_long) + // gauge reads
		   num_dir*2*spinor_bytes +                       // spinor reads
		   spinor_bytes)*sites;                           // spinor write
	  if (x) bytes += spinor_bytes;

	  if (dslashParam.kernel_type == KERNEL_POLICY) break;
	  // now correct for bytes done by exterior kernel
	  long long ghost_sites = 0;
	  for (int d=0; d<4; d++) if (dslashParam.commDim[d]) ghost_sites += 2*in->GhostFace()[d];
	  bytes -= ghost_bytes * ghost_sites;
	  
	  break;
	}
      }
      return bytes;
    }

  };
#endif // GPU_STAGGERED_DIRAC

  void improvedStaggeredDslashCuda(cudaColorSpinorField *out, const cudaGaugeField &fatGauge, 
				   const cudaGaugeField &longGauge, const cudaColorSpinorField *in,
				   const int parity, const int dagger, const cudaColorSpinorField *x,
				   const double &k, const int *commOverride, TimeProfile &profile)
  {
#if (defined GPU_STAGGERED_DIRAC && defined USE_LEGACY_DSLASH)
    const_cast<cudaColorSpinorField*>(in)->createComms(3);

    DslashCuda *dslash = nullptr;
    if (in->Precision() == QUDA_DOUBLE_PRECISION) {
      dslash = new StaggeredDslashCuda<double2, double2, double2, double>
        (out, fatGauge, longGauge, in, x, k, parity, dagger, commOverride);
    } else if (in->Precision() == QUDA_SINGLE_PRECISION) {
      dslash = new StaggeredDslashCuda<float2, float2, float4, float>
	(out, fatGauge, longGauge, in, x, k, parity, dagger, commOverride);
    } else if (in->Precision() == QUDA_HALF_PRECISION) {	
      dslash = new StaggeredDslashCuda<short2, short2, short4, short>
	(out, fatGauge, longGauge, in, x, k, parity, dagger, commOverride);
    }

    // the parameters passed to dslashCuda must be 4-d volume and 3-d
    // faces because Ls is added as the y-dimension in thread space
    int ghostFace[QUDA_MAX_DIM];
    for (int i=0; i<4; i++) ghostFace[i] = in->GhostFace()[i] / in->X(4);

    dslash::DslashPolicyTune<DslashCuda> dslash_policy(
        *dslash, const_cast<cudaColorSpinorField *>(in), in->Volume() / in->X(4), ghostFace, profile);
    dslash_policy.apply(0);

    delete dslash;
#else
    errorQuda("Staggered dslash has not been built");
#endif  // GPU_STAGGERED_DIRAC
  }

}
